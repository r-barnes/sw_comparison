#include "hip/hip_runtime.h"
/***************************************************************************
 *   Copyright (C) 2006                                                    *
 *                                                                         *
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 2 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 *   This program is distributed in the hope that it will be useful,       *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with this program; if not, write to the                         *
 *   Free Software Foundation, Inc.,                                       *
 *   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 ***************************************************************************/

/* 
 * Device code.
 */

#ifndef _TEMPLATE_KERNEL2_H_
#define _TEMPLATE_KERNEL2_H_


#define HSH( index )      CUT_BANK_CHECKER(Hsh, (index))
#define ESH( index )      CUT_BANK_CHECKER(Esh, (index))
#define FSH( index )      CUT_BANK_CHECKER(Fsh, (index))


#include "sbtmatrix.h"
#include "swutils.h"

/*
////////////////////////////////////////////////////////////////////////////////
// Simple test kernel for device functionality
////////////////////////////////////////////////////////////////////////////////
__global__ void smithwatermanKernel_last63antidiag( const char* g_seqlib, unsigned totBytesUsed, unsigned numSeqs, unsigned 									seqOffset, unsigned *g_offsets, unsigned *g_sizes, unsigned alpha, unsigned beta,
									int *g_Hdata, int *g_Edata, int *g_Fdata, int *g_scores) 
{
	const unsigned int NUM_THREADS = 64;

  
	// shared memory

	// 2 previous of H diagonals need to be recorded for the calculation
	// one initial 0 is always needed for border conditions
	// the last position of H is needed for max current scores
	__shared__  int Hsh[NUM_THREADS*2+3];
	__shared__  int Esh[NUM_THREADS+1];
	__shared__  int Fsh[NUM_THREADS+1];


	// access thread id and block id
	const unsigned int tid = threadIdx.x;
	const unsigned int blid = blockIdx.x;

	// read the substitution matrix
	loadPAM_64threads(tid);

	unsigned sizeA = g_sizes[blid+seqOffset];
	unsigned sizeB = g_sizes[0];

	// read in input data from global memory
	// use the bank checker macro to check for bank conflicts during host
	// emulation
	unsigned libOffset = g_offsets[blid+seqOffset];
	
	ASEQ64(0) = g_seqlib[ libOffset ];
	ASEQ64(tid+1) = g_seqlib[ libOffset + sizeA - NUM_THREADS + tid ];

	BSEQ64(0) = g_seqlib[0]; // the one we search for
	BSEQ64(tid+1) = g_seqlib[sizeB - NUM_THREADS + tid ]; // the one we search for
	
	HSH(tid+1) = g_Hdata[(2*(blid+seqOffset))*NUM_THREADS + tid]; 
	HSH(NUM_THREADS+1 + tid + 1) = g_Hdata[(2*(blid+seqOffset))*NUM_THREADS + NUM_THREADS + tid];

	ESH(tid+1) = g_Edata[(blid+seqOffset)*NUM_THREADS + tid];
	FSH(tid+1) = g_Fdata[(blid+seqOffset)*NUM_THREADS + tid]; 
	
	HSH(0) = 0; 
	HSH(NUM_THREADS+1) = 0;
	ESH(0) = 0; 
	FSH(0) = 0; 
	HSH(NUM_THREADS*2+2) = g_scores[blid+seqOffset];

	unsigned lastDiag = 0; // switches between 0 and 1
	unsigned curDiagIdx = 0; 
		

	// la prima antidiagonale e' quella che per prima esclude un carattere delle due sequenze
	// to be curDiagIdx < 63
	for (; curDiagIdx < 63; ++curDiagIdx) {
		__syncthreads();

		int h=0,e=0,f=0;

		swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

		lastDiag = 1 - lastDiag;
	
		__syncthreads();
		HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
		ESH(tid + 1) = e;
		FSH(tid + 1) = f;

		//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

		SCORES(tid) = h;
		//SCORES(NUM_THREADS + tid) = 0;
		__syncthreads();

		max64( tid, HSH(NUM_THREADS*2+2) );
	}	

	// write data to global memory
	__syncthreads();
	g_Hdata[(2*(blid+seqOffset))*NUM_THREADS + tid] = HSH(lastDiag*(NUM_THREADS+1) + tid + 1);
	g_Hdata[(2*(blid+seqOffset))*NUM_THREADS + NUM_THREADS + tid] = HSH( (1-lastDiag)*(NUM_THREADS+1) + tid + 1);
	g_Edata[(blid+seqOffset)*NUM_THREADS + tid] = ESH(tid + 1);
	g_Fdata[(blid+seqOffset)*NUM_THREADS + tid] = FSH(tid + 1);
	g_scores[blid+seqOffset] = HSH(NUM_THREADS*2+2);

}
*/

__global__ void smithwatermanKernel_last63antidiag( const char* g_strToAlign, const char* g_seqlib, unsigned totBytesUsed, unsigned numSeqs, unsigned seqOffset, unsigned *g_offsets, unsigned *g_sizes, unsigned alpha, unsigned beta, int *g_Hdata, int *g_Edata, int *g_Fdata, int *g_scores) 
{
	const unsigned int NUM_THREADS = 64;

  
	// shared memory

	// 2 previous of H diagonals need to be recorded for the calculation
	// one initial 0 is always needed for border conditions
	// the last position of H is needed for max current scores
	__shared__  int Hsh[NUM_THREADS*2+3];
	__shared__  int Esh[NUM_THREADS+1];
	__shared__  int Fsh[NUM_THREADS+1];


	// access thread id and block id
	const unsigned int tid = threadIdx.x;
	const unsigned int blid = blockIdx.x;

	// read the substitution matrix
	loadPAM_64threads(tid);

	unsigned sizeA = g_sizes[blid+seqOffset];
	unsigned sizeB = g_sizes[0];

	// read in input data from global memory
	// use the bank checker macro to check for bank conflicts during host
	// emulation
	unsigned libOffset = g_offsets[blid+seqOffset];

	ASEQ64(0) = g_seqlib[ libOffset ];
	ASEQ64(tid+1) = g_seqlib[ libOffset + sizeA - NUM_THREADS + tid ];

	BSEQ64(0) = g_strToAlign[0]; // the one we search for
	BSEQ64(tid+1) = g_strToAlign[sizeB - NUM_THREADS + tid ]; // the one we search for
	
	HSH(tid+1) = g_Hdata[(2*(blid+seqOffset))*NUM_THREADS + tid]; 
	HSH(NUM_THREADS+1 + tid + 1) = g_Hdata[(2*(blid+seqOffset))*NUM_THREADS + NUM_THREADS + tid];

	ESH(tid+1) = g_Edata[(blid+seqOffset)*NUM_THREADS + tid];
	FSH(tid+1) = g_Fdata[(blid+seqOffset)*NUM_THREADS + tid]; 
	
	HSH(0) = 0; 
	HSH(NUM_THREADS+1) = 0;
	ESH(0) = 0; 
	FSH(0) = 0; 
	HSH(NUM_THREADS*2+2) = g_scores[blid+seqOffset];

	unsigned lastDiag = 0; // switches between 0 and 1
	unsigned curDiagIdx = 0;
	int tempIdx = 63 - tid;
		

	// loop unrolling curDiagIdx < 63
	//____________________________________________RUN_1____________________________________________
	//curDiagIdx = 0
	__syncthreads();

	int h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_1________________________________________


	//____________________________________________RUN_2____________________________________________
	curDiagIdx = 1;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_2________________________________________


	//____________________________________________RUN_3____________________________________________
	curDiagIdx = 2;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_3________________________________________

	//____________________________________________RUN_4____________________________________________
	curDiagIdx = 3;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_4________________________________________

	//____________________________________________RUN_5____________________________________________
	curDiagIdx = 4;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_5________________________________________

	//____________________________________________RUN_6____________________________________________
	curDiagIdx = 5;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_6________________________________________

	//____________________________________________RUN_7____________________________________________
	curDiagIdx = 6;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_7________________________________________

	//____________________________________________RUN_8____________________________________________
	curDiagIdx = 7;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_8________________________________________

	//____________________________________________RUN_9____________________________________________
	curDiagIdx = 8;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_9________________________________________

	//____________________________________________RUN_10____________________________________________
	curDiagIdx = 9;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_10________________________________________

	//____________________________________________RUN_11____________________________________________
	curDiagIdx = 10;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_11________________________________________

	//____________________________________________RUN_12____________________________________________
	curDiagIdx = 11;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_12________________________________________

	//____________________________________________RUN_13____________________________________________
	curDiagIdx = 12;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_13________________________________________


	//____________________________________________RUN_14____________________________________________
	curDiagIdx = 13;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_14________________________________________

	//____________________________________________RUN_15____________________________________________
	curDiagIdx = 14;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_15________________________________________

	//____________________________________________RUN_16____________________________________________
	curDiagIdx = 15;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_16________________________________________

	//____________________________________________RUN_17____________________________________________
	curDiagIdx = 16;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_17________________________________________

	//____________________________________________RUN_18____________________________________________
	curDiagIdx = 17;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_18________________________________________

	//____________________________________________RUN_19____________________________________________
	curDiagIdx = 18;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_19________________________________________

	//____________________________________________RUN_20____________________________________________
	curDiagIdx = 19;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_20________________________________________

	//____________________________________________RUN_21____________________________________________
	curDiagIdx = 20;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_21________________________________________

	//____________________________________________RUN_22____________________________________________
	curDiagIdx = 21;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_22________________________________________

	//____________________________________________RUN_23____________________________________________
	curDiagIdx = 22;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_23________________________________________

	//____________________________________________RUN_24____________________________________________
	curDiagIdx = 23;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_24________________________________________

	//____________________________________________RUN_25____________________________________________
	curDiagIdx = 24;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_25________________________________________

	//____________________________________________RUN_26____________________________________________
	curDiagIdx = 25;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_26________________________________________

	//____________________________________________RUN_27____________________________________________
	curDiagIdx = 26;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_27________________________________________

	//____________________________________________RUN_28____________________________________________
	curDiagIdx = 27;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_28________________________________________

	//____________________________________________RUN_29____________________________________________
	curDiagIdx = 28;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_29________________________________________

	//____________________________________________RUN_30____________________________________________
	curDiagIdx = 29;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_30________________________________________

	//____________________________________________RUN_31____________________________________________
	curDiagIdx = 30;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tid) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max64( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_31________________________________________


	//____________________________________________RUN_32____________________________________________
	curDiagIdx = 31;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max32( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_32________________________________________


	//____________________________________________RUN_33____________________________________________
	curDiagIdx = 32;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max32( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_33________________________________________


	//____________________________________________RUN_34____________________________________________
	curDiagIdx = 33;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max32( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_34________________________________________


	//____________________________________________RUN_35____________________________________________
	curDiagIdx = 34;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max32( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_35________________________________________


	//____________________________________________RUN_36____________________________________________
	curDiagIdx = 35;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max32( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_36________________________________________


	//____________________________________________RUN_37____________________________________________
	curDiagIdx = 36;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max32( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_37________________________________________


	//____________________________________________RUN_38____________________________________________
	curDiagIdx = 37;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max32( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_38________________________________________


	//____________________________________________RUN_39____________________________________________
	curDiagIdx = 38;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max32( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_39________________________________________


	//____________________________________________RUN_40____________________________________________
	curDiagIdx = 39;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max32( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_40________________________________________


	//____________________________________________RUN_41____________________________________________
	curDiagIdx = 40;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max32( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_41________________________________________


	//____________________________________________RUN_42____________________________________________
	curDiagIdx = 41;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max32( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_42________________________________________


	//____________________________________________RUN_43____________________________________________
	curDiagIdx = 42;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max32( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_43________________________________________


	//____________________________________________RUN_44____________________________________________
	curDiagIdx = 43;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max32( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_44________________________________________


	//____________________________________________RUN_45____________________________________________
	curDiagIdx = 44;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max32( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_45________________________________________


	//____________________________________________RUN_46____________________________________________
	curDiagIdx = 45;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max32( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_46________________________________________


	//____________________________________________RUN_47____________________________________________
	curDiagIdx = 46;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max32( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_47________________________________________


	//____________________________________________RUN_48____________________________________________
	curDiagIdx = 47;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max16( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_48________________________________________
	

	//____________________________________________RUN_49____________________________________________
	curDiagIdx = 48;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max16( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_49________________________________________

	//____________________________________________RUN_50____________________________________________
	curDiagIdx = 49;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max16( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_50________________________________________

	//____________________________________________RUN_51____________________________________________
	curDiagIdx = 50;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max16( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_51________________________________________

	//____________________________________________RUN_52____________________________________________
	curDiagIdx = 51;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max16( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_52________________________________________

	//____________________________________________RUN_53____________________________________________
	curDiagIdx = 52;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max16( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_53________________________________________


	//____________________________________________RUN_54____________________________________________
	curDiagIdx = 53;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max16( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_54________________________________________

	//____________________________________________RUN_55____________________________________________
	curDiagIdx = 54;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max16( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_55________________________________________

	//____________________________________________RUN_56____________________________________________
	curDiagIdx = 55;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max8( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_56________________________________________
	

	//____________________________________________RUN_57____________________________________________
	curDiagIdx = 56;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max8( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_57________________________________________

	//____________________________________________RUN_58____________________________________________
	curDiagIdx = 57;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max8( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_58________________________________________

	//____________________________________________RUN_59____________________________________________
	curDiagIdx = 58;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max8( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_59________________________________________

	//____________________________________________RUN_60____________________________________________
	curDiagIdx = 59;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max4( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_60________________________________________


	//____________________________________________RUN_61____________________________________________
	curDiagIdx = 60;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	SCORES(tempIdx) = h;
	//SCORES(NUM_THREADS + tid) = 0;
	__syncthreads();

	max4( tid, HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_61________________________________________


	//____________________________________________RUN_62____________________________________________
	curDiagIdx = 61;

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	__syncthreads();

	HSH(NUM_THREADS*2+2) = max( HSH(lastDiag*(NUM_THREADS+1) + 63), HSH(NUM_THREADS*2+2) );
	HSH(NUM_THREADS*2+2) = max( HSH(lastDiag*(NUM_THREADS+1) + 64), HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_62________________________________________


	//____________________________________________RUN_63____________________________________________
	curDiagIdx = 62;
	__syncthreads();

	h=0,e=0,f=0;

	swcalc_downright(curDiagIdx, sizeA, sizeB, HSH( lastDiag*(NUM_THREADS+1) + tid ), FSH( tid ), HSH( lastDiag*(NUM_THREADS+1)+ tid+ 1 ), ESH( tid + 1 ), HSH( (1 - lastDiag)*(NUM_THREADS+1) + tid  ), alpha, beta, h, e, f);

	lastDiag = 1 - lastDiag;
	
	__syncthreads();
	HSH(lastDiag*(NUM_THREADS+1) + tid + 1) = h;
	ESH(tid + 1) = e;
	FSH(tid + 1) = f;

	//if (ASEQ(idxSeqA) != '[' && BSEQ(idxSeqB) != '[') printf("tid=%u diag=%u %c-%c res=%d\t\t", tid, curDiagIdx, ASEQ(idxSeqA), BSEQ(idxSeqB), pam[ASEQ(idxSeqA)-60][BSEQ(idxSeqB)-60]);

	__syncthreads();

	HSH(NUM_THREADS*2+2) = max( HSH(lastDiag*(NUM_THREADS+1) + 64), HSH(NUM_THREADS*2+2) );

	//________________________________________END_OF_RUN_63________________________________________


	// write data to global memory
	__syncthreads();
	g_Hdata[(2*(blid+seqOffset))*NUM_THREADS + tid] = HSH(lastDiag*(NUM_THREADS+1) + tid + 1);
	g_Hdata[(2*(blid+seqOffset))*NUM_THREADS + NUM_THREADS + tid] = HSH( (1-lastDiag)*(NUM_THREADS+1) + tid + 1);
	g_Edata[(blid+seqOffset)*NUM_THREADS + tid] = ESH(tid + 1);
	g_Fdata[(blid+seqOffset)*NUM_THREADS + tid] = FSH(tid + 1);
	g_scores[blid+seqOffset] = HSH(NUM_THREADS*2+2);

}


#endif // #ifndef _TEMPLATE_KERNEL2_H_

