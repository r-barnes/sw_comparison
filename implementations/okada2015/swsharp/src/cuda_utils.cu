/*
swsharp - CUDA parallelized Smith Waterman with applying Hirschberg's and 
Ukkonen's algorithm and dynamic cell pruning.
Copyright (C) 2013 Matija Korpar, contributor Mile Šikić

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

Contact the author by mkorpar@gmail.com.
*/

#include <stdlib.h>
#include <string.h>

#include <swsharp/error.h>
#include <swsharp/utils.h>
#include <swsharp/cuda_utils.h>

extern void cudaGetCards(int** cards, int* cardsLen) {

#ifdef __HIPCC__
    hipGetDeviceCount(cardsLen);
    
    *cards = (int*) malloc(*cardsLen * sizeof(int));
    
    for (int i = 0; i < *cardsLen; ++i) {
        (*cards)[i] = i;   
    }
#else
    *cards = NULL;
    *cardsLen = 0;
#endif
}

extern int cudaCheckCards(int* cards, int cardsLen) {

#ifdef __HIPCC__
    int maxDeviceId;
    hipGetDeviceCount(&maxDeviceId);
    
    for (int i = 0; i < cardsLen; ++i) {
        if (cards[i] >= maxDeviceId) {
            return 0;
        }   
    }
    
    return 1;
#else
    return cardsLen == 0;
#endif
}

extern size_t cudaMinimalGlobalMemory(int* cards, int cardsLen) {

#ifdef __HIPCC__

    if (cards == NULL || cardsLen == 0) {
        return 0;
    }

    size_t minMem = (size_t) -1;
    for (int i = 0; i < cardsLen; ++i) {

        hipDeviceProp_t cdp;
        hipGetDeviceProperties(&cdp, i);

        minMem = MIN(minMem, cdp.totalGlobalMem);
    }

    return minMem;
#else
    return 0;
#endif
}

