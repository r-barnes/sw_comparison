#include "hip/hip_runtime.h"
/** maximum X per block (used in dimensions for blocks and amount of shared memory */
#define SHARED_X ${SHARED_X}
/** maximum Y per block (used in dimensions for blocks and amount of shared memory */
#define SHARED_Y ${SHARED_Y}

/** kernel contains a for-loop in which the score is calculated. */
#define DIAGONAL SHARED_X + SHARED_Y

/** amount of score elements in a single block */
#define blockSize (SHARED_X * SHARED_Y)

/** amount of blocks across the X axis */
#define XdivSHARED_X (X/SHARED_X)
/** amount of blocks across the Y axis */
#define YdivSHARED_Y (Y/SHARED_Y)

/** character used to fill the sequence if length < X */
#define FILL_CHARACTER '\0'
#define FILL_SCORE -1E10f

/** this value is used to allocate enough memory to store the starting points */
#define MAXIMUM_NUMBER_STARTING_POINTS (NUMBER_SEQUENCES*NUMBER_TARGETS*1000)

/**** Other definitions ****/

/** bit mask to get the negative value of a float, or to keep it negative */
#define SIGN_BIT_MASK 0x80000000
#define MAX_LINE_LENGTH 500
#define AFFINE_GAP_INIT -1E10f

/* Scorings matrix for each thread block */
typedef struct {
    float value[SHARED_X][SHARED_Y];
}  LocalMatrix;

/* Scorings matrix for each sequence alignment */
typedef struct {
    LocalMatrix matrix[XdivSHARED_X][YdivSHARED_Y];
} ScoringsMatrix;

/* Scorings matrix for entire application */
typedef struct {
    ScoringsMatrix metaMatrix[NUMBER_SEQUENCES][NUMBER_TARGETS];
} GlobalMatrix;

typedef struct {
    float value[XdivSHARED_X][YdivSHARED_Y];
} BlockMaxima;

typedef struct {
    BlockMaxima blockMaxima[NUMBER_SEQUENCES][NUMBER_TARGETS];
} GlobalMaxima;

typedef struct {
    unsigned char value[SHARED_X][SHARED_Y];
} LocalDirection;

typedef struct {
    LocalDirection localDirection[XdivSHARED_X][YdivSHARED_Y];
} Direction;

typedef struct {
    Direction direction[NUMBER_SEQUENCES][NUMBER_TARGETS];
} GlobalDirection;

typedef struct {
    unsigned int sequence;
    unsigned int target;
    unsigned int blockX;
    unsigned int blockY;
    unsigned int valueX;
    unsigned int valueY;
    float score;
    float maxScore;
    float posScore;
} StartingPoint;

typedef struct {
    StartingPoint startingPoint[MAXIMUM_NUMBER_STARTING_POINTS];
} StartingPoints;

typedef struct {
    float value[NUMBER_TARGETS];
} TargetMaxima;

typedef struct {
    float value[NUMBER_SEQUENCES];
} SequenceMaxima;

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <builtin_types.h>
#include <time.h>

/**
 * The calculateScore function checks the alignment per block. It calculates the score for each cell in
 * shared memory.
 * @matrix   The scorings matrix
 * @x        The start x block position in the alignment to be calculated
 * @y        The start y block position in the alignment to be calculated
 * @numberOfBlocks The amount of blocks within an alignment which can be calculated
 * @seq1     The upper sequence in the alignment
 * @seq2     The left sequence in the alignment
 */
extern "C"
__global__ void calculateScore(GlobalMatrix *matrix, unsigned int x, unsigned int y, unsigned int numberOfBlocks, char *sequences, char *targets, GlobalMaxima *globalMaxima, GlobalDirection *globalDirection);

extern "C"
__global__ void calculateScoreAffineGap(GlobalMatrix *matrix, GlobalMatrix *matrix_i, GlobalMatrix *matrix_j, unsigned int x, unsigned int y, unsigned int numberOfBlocks, char *sequences, char *targets, GlobalMaxima *globalMaxima, GlobalDirection *globalDirection);

extern "C"
__global__ void traceback(GlobalMatrix *matrix, unsigned int x, unsigned int y, unsigned int numberOfBlocks, GlobalMaxima *globalMaxima, GlobalDirection *globalDirection, GlobalDirection *globalDirectionZeroCopy, unsigned int *indexIncrement, StartingPoints *startingPoints, float *maxPossibleScore);

extern "C"
__global__ void tracebackAffineGap(GlobalMatrix *matrix, GlobalMatrix *matrix_i, GlobalMatrix *matrix_j, unsigned int x, unsigned int y, unsigned int numberOfBlocks, GlobalMaxima *globalMaxima, GlobalDirection *globalDirection, GlobalDirection *globalDirectionZeroCopy, unsigned int *indexIncrement, StartingPoints *startingPoints, float *maxPossibleScore);


/**
 * The calculateScore function checks the alignment per block. It calculates the score for each cell in
 * shared memory
 * @matrix   The scorings matrix
 * @x        The start x block position in the alignment to be calculated
 * @y        The start y block position in the alignment to be calculated
 * @numberOfBlocks The amount of blocks within an alignment which can be calculated
 * @seq1     The upper sequence in the alignment
 * @seq2     The left sequence in the alignment
 */
__global__ void calculateScore(
        GlobalMatrix *matrix, unsigned int x, unsigned int y, unsigned int numberOfBlocks,
        char *sequences, char *targets,
        GlobalMaxima *globalMaxima,
        GlobalDirection *globalDirection
        ) {
    /**
     * shared memory block for calculations. It requires
     * extra (+1 in both directions) space to hold
     * Neighboring cells
     */
    __shared__ float s_matrix[SHARED_X+1][SHARED_Y+1];
    /**
     * shared memory block for storing the maximum value of each neighboring cell.
     * Careful: the s_maxima[SHARED_X][SHARED_Y] does not contain the maximum value
     * after the calculation loop! This value is determined at the end of this
     * function.
     */
    __shared__ float s_maxima[SHARED_X][SHARED_Y];

    // calculate indices:
    //unsigned int yDIVnumSeq = (blockIdx.y/NUMBER_SEQUENCES);
    unsigned int blockx = x - blockIdx.y/NUMBER_TARGETS;//yDIVnumSeq;
    unsigned int blocky = y + blockIdx.y/NUMBER_TARGETS;//yDIVnumSeq;
    unsigned int tIDx = threadIdx.x;
    unsigned int tIDy = threadIdx.y;
    unsigned int bIDx = blockIdx.x;
    unsigned int bIDy = blockIdx.y%NUMBER_TARGETS;///numberOfBlocks;
    unsigned char direction = NO_DIRECTION;


    // indices of the current characters in both sequences.
    int seqIndex1 = tIDx + bIDx * X + blockx * SHARED_X;
    int seqIndex2 = tIDy + bIDy * Y + blocky * SHARED_Y;


    /* the next block is to get the maximum value from surrounding blocks. This maximum values is compared to the
     * first element in the shared score matrix s_matrix.
     */
    float maxPrev = 0.0f;
    if (!tIDx && !tIDy) {
        if (blockx && blocky) {
            maxPrev = max(max(globalMaxima->blockMaxima[bIDx][bIDy].value[blockx-1][blocky-1], globalMaxima->blockMaxima[bIDx][bIDy].value[blockx-1][blocky]), globalMaxima->blockMaxima[bIDx][bIDy].value[blockx][blocky-1]);
        }
        else if (blockx) {
            maxPrev = globalMaxima->blockMaxima[bIDx][bIDy].value[blockx-1][blocky];
        }
        else if (blocky) {
            maxPrev = globalMaxima->blockMaxima[bIDx][bIDy].value[blockx][blocky-1];
        }
    }
    // local scorings variables:
    float currentScore, ulS, lS, uS;
    float innerScore = 0.0f;
    /**
     * tXM1 and tYM1 are to store the current value of the thread Index. tIDx and tIDy are
     * both increased with 1 later on.
     */
    unsigned int tXM1 = tIDx;
    unsigned int tYM1 = tIDy;

    // shared location for the parts of the 2 sequences, for faster retrieval later on:
    __shared__ char s_seq1[SHARED_X];
    __shared__ char s_seq2[SHARED_Y];

    // copy sequence data to shared memory (shared is much faster than global)
    if (!tIDy)
        s_seq1[tIDx] = sequences[seqIndex1];
    if (!tIDx)
        s_seq2[tIDy] = targets[seqIndex2];

    // set both matrices to zero
    s_matrix[tIDx][tIDy] = 0.0f;
    s_maxima[tIDx][tIDy] = 0.0f;

    if (tIDx == SHARED_X-1  && ! tIDy)
        s_matrix[SHARED_X][0] = 0.0f;
    if (tIDy == SHARED_Y-1  && ! tIDx)
        s_matrix[0][SHARED_Y] = 0.0f;

    /**** sync barrier ****/
    s_matrix[tIDx][tIDy] = 0.0f;
    __syncthreads();

    // initialize outer parts of the matrix:
    if (!tIDx || !tIDy) {
        if (tIDx == SHARED_X-1)
            s_matrix[tIDx+1][tIDy] = 0.0f;
        if (tIDy == SHARED_Y-1)
            s_matrix[tIDx][tIDy+1] = 0.0f;
        if (blockx && !tIDx) {
            s_matrix[0][tIDy+1] = (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky].value[SHARED_X-1][tIDy];
        }
        if (blocky && !tIDy) {
            s_matrix[tIDx+1][0] = (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx][blocky-1].value[tIDx][SHARED_Y-1];
        }
        if (blockx && blocky && !tIDx && !tIDy){
            s_matrix[0][0] = (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky-1].value[SHARED_X-1][SHARED_Y-1];
        }
    }
    // set inner score (aka sequence match/mismatch score):
    char charS1 = s_seq1[tIDx];
    char charS2 = s_seq2[tIDy];

    innerScore = charS1 == FILL_CHARACTER || charS2 == FILL_CHARACTER ? FILL_SCORE : scoringsMatrix[charS1-characterOffset][charS2-characterOffset];

    // transpose the index
    ++tIDx;
    ++tIDy;
    // set shared matrix to zero (starting point!)
    s_matrix[tIDx][tIDy] = 0.0f;


    // wait until all elements have been copied to the shared memory block
        /**** sync barrier ****/
    __syncthreads();

    currentScore = 0.0f;

    for (int i=0; i < DIAGONAL; ++i) {
        if (i == tXM1+ tYM1) {
            // calculate only when there are two valid characters
            // this is necessary when the two sequences are not of equal length
            // this is the SW-scoring of the cell:

          ulS = s_matrix[tXM1][tYM1] + innerScore;
          lS = s_matrix[tXM1][tIDy] + gapScore;
          uS = s_matrix[tIDx][tYM1] + gapScore;

            if (currentScore < lS) { // score comes from left
                currentScore = lS;
                direction = LEFT_DIRECTION;
            }
            if (currentScore < uS) { // score comes from above
                currentScore = uS;
                direction = UPPER_DIRECTION;
            }
            if (currentScore < ulS) { // score comes from upper left
                currentScore = ulS;
                direction = UPPER_LEFT_DIRECTION;
            }
            s_matrix[tIDx][tIDy] = innerScore == FILL_SCORE ? 0.0 : currentScore; // copy score to matrix
        }

        else if (i-1 == tXM1 + tYM1 ){
                // use this to find max
            if (i==1) {
                s_maxima[0][0] = max(maxPrev, currentScore);
            }
            else if (!tXM1 && tYM1) {
                s_maxima[0][tYM1] = max(s_maxima[0][tYM1-1], currentScore);
            }
            else if (!tYM1 && tXM1) {
                s_maxima[tXM1][0] = max(s_maxima[tXM1-1][0], currentScore);
            }
            else if (tXM1 && tYM1 ){
                s_maxima[tXM1][tYM1] = max(s_maxima[tXM1-1][tYM1], max(s_maxima[tXM1][tYM1-1], currentScore));
            }
        }
        // wait until all threads have calculated their new score
            /**** sync barrier ****/
        __syncthreads();
    }
    // copy end score to the scorings matrix:
    (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx][blocky].value[tXM1][tYM1] = s_matrix[tIDx][tIDy];
    (*globalDirection).direction[bIDx][bIDy].localDirection[blockx][blocky].value[tXM1][tYM1] = direction;

    if (tIDx==SHARED_X && tIDy==SHARED_Y)
        globalMaxima->blockMaxima[bIDx][bIDy].value[blockx][blocky] = max(currentScore, max(s_maxima[SHARED_X-2][SHARED_Y-1], s_maxima[SHARED_X-1][SHARED_Y-2]));

    // wait until all threads have copied their score:
        /**** sync barrier ****/
    __syncthreads();
}

/**
 * The calculateScoreAffineGape function checks the alignment per block. It calculates the score for each cell in
 * shared memory using the affine gap penalty score
 * @matrix   The scorings matrix
 * @matrix_i The scorings matrix, gaps in X
 * @matrix_j The scorings matrix, gaps in Y
 * @x        The start x block position in the alignment to be calculated
 * @y        The start y block position in the alignment to be calculated
 * @numberOfBlocks The amount of blocks within an alignment which can be calculated
 * @seq1     The upper sequence in the alignment
 * @seq2     The left sequence in the alignment
 */
__global__ void calculateScoreAffineGap(GlobalMatrix *matrix, GlobalMatrix *matrix_i, GlobalMatrix *matrix_j,
		unsigned int x, unsigned int y, unsigned int numberOfBlocks,
        char *sequences, char *targets,
        GlobalMaxima *globalMaxima,
        GlobalDirection *globalDirection
        ) {
    /**
     * shared memory block for calculations. It requires
     * extra (+1 in both directions) space to hold
     * Neighboring cells
     */
    __shared__ float s_matrix[SHARED_X+1][SHARED_Y+1];
    __shared__ float s_matrix_i[SHARED_X+1][SHARED_Y+1];
    __shared__ float s_matrix_j[SHARED_X+1][SHARED_Y+1];
    /**
     * shared memory block for storing the maximum value of each neighboring cell.
     * Careful: the s_maxima[SHARED_X][SHARED_Y] does not contain the maximum value
     * after the calculation loop! This value is determined at the end of this
     * function.
     */
    __shared__ float s_maxima[SHARED_X][SHARED_Y];

    // calculate indices:
    //unsigned int yDIVnumSeq = (blockIdx.y/NUMBER_SEQUENCES);
    unsigned int blockx = x - blockIdx.y/NUMBER_TARGETS;//yDIVnumSeq;
    unsigned int blocky = y + blockIdx.y/NUMBER_TARGETS;//yDIVnumSeq;
    unsigned int tIDx = threadIdx.x;
    unsigned int tIDy = threadIdx.y;
    unsigned int bIDx = blockIdx.x;
    unsigned int bIDy = blockIdx.y%NUMBER_TARGETS;///numberOfBlocks;
    unsigned char direction = NO_DIRECTION;
    unsigned char direction_i = NO_DIRECTION;
    unsigned char direction_j = NO_DIRECTION;



    // indices of the current characters in both sequences.
    int seqIndex1 = tIDx + bIDx * X + blockx * SHARED_X;
    int seqIndex2 = tIDy + bIDy * Y + blocky * SHARED_Y;


    /* the next block is to get the maximum value from surrounding blocks. This maximum values is compared to the
     * first element in the shared score matrix s_matrix.
     */
    float maxPrev = 0.0f;
    if (!tIDx && !tIDy) {
        if (blockx && blocky) {
            maxPrev = max(max(globalMaxima->blockMaxima[bIDx][bIDy].value[blockx-1][blocky-1], globalMaxima->blockMaxima[bIDx][bIDy].value[blockx-1][blocky]), globalMaxima->blockMaxima[bIDx][bIDy].value[blockx][blocky-1]);
        }
        else if (blockx) {
            maxPrev = globalMaxima->blockMaxima[bIDx][bIDy].value[blockx-1][blocky];
        }
        else if (blocky) {
            maxPrev = globalMaxima->blockMaxima[bIDx][bIDy].value[blockx][blocky-1];
        }
    }
    // local scorings variables:
    float currentScore,currentScore_i, currentScore_j, m_M, m_I, m_J;
    float innerScore = 0.0f;
    /**
     * tXM1 and tYM1 are to store the current value of the thread Index. tIDx and tIDy are
     * both increased with 1 later on.
     */
    unsigned int tXM1 = tIDx;
    unsigned int tYM1 = tIDy;

    // shared location for the parts of the 2 sequences, for faster retrieval later on:
    __shared__ char s_seq1[SHARED_X];
    __shared__ char s_seq2[SHARED_Y];

    // copy sequence data to shared memory (shared is much faster than global)
    if (!tIDy)
        s_seq1[tIDx] = sequences[seqIndex1];
    if (!tIDx)
        s_seq2[tIDy] = targets[seqIndex2];
    // set matrices to zero
    s_matrix[tIDx][tIDy] = 0.0f;
    s_matrix_i[tIDx][tIDy] = AFFINE_GAP_INIT;
    s_matrix_j[tIDx][tIDy] = AFFINE_GAP_INIT;
    s_maxima[tIDx][tIDy] = 0.0f;

    if (tIDx == SHARED_X-1  && ! tIDy) {
        s_matrix[SHARED_X][0] = 0.0f;
        s_matrix_i[SHARED_X][0] = AFFINE_GAP_INIT;
        s_matrix_j[SHARED_X][0] = AFFINE_GAP_INIT;
    }
    if (tIDy == SHARED_Y-1  && ! tIDx) {
        s_matrix[0][SHARED_Y] = 0.0f;
        s_matrix_i[0][SHARED_Y] = AFFINE_GAP_INIT;
        s_matrix_j[0][SHARED_Y] = AFFINE_GAP_INIT;
    }

    /**** sync barrier ****/
    __syncthreads();

    // initialize outer parts of the matrix:
    if (!tIDx || !tIDy) {
        if (tIDx == SHARED_X-1) {
            s_matrix[tIDx+1][tIDy] = 0.0f;
            s_matrix_i[tIDx+1][tIDy] = AFFINE_GAP_INIT;
            s_matrix_j[tIDx+1][tIDy] = AFFINE_GAP_INIT;
        }
        if (tIDy == SHARED_Y-1) {
            s_matrix[tIDx][tIDy+1] = 0.0f;
            s_matrix_i[tIDx][tIDy+1] = AFFINE_GAP_INIT;
            s_matrix_j[tIDx][tIDy+1] = AFFINE_GAP_INIT;
        }
        if (blockx && !tIDx) {
            s_matrix[0][tIDy+1] = (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky].value[SHARED_X-1][tIDy];
            s_matrix_i[0][tIDy+1] = (*matrix_i).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky].value[SHARED_X-1][tIDy];
            s_matrix_j[0][tIDy+1] = (*matrix_j).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky].value[SHARED_X-1][tIDy];
        }
        if (blocky && !tIDy) {
            s_matrix[tIDx+1][0] = (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx][blocky-1].value[tIDx][SHARED_Y-1];
            s_matrix_i[tIDx+1][0] = (*matrix_i).metaMatrix[bIDx][bIDy].matrix[blockx][blocky-1].value[tIDx][SHARED_Y-1];
            s_matrix_j[tIDx+1][0] = (*matrix_j).metaMatrix[bIDx][bIDy].matrix[blockx][blocky-1].value[tIDx][SHARED_Y-1];
        }
        if (blockx && blocky && !tIDx && !tIDy){
            s_matrix[0][0] = (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky-1].value[SHARED_X-1][SHARED_Y-1];
            s_matrix_i[0][0] = (*matrix_i).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky-1].value[SHARED_X-1][SHARED_Y-1];
            s_matrix_j[0][0] = (*matrix_j).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky-1].value[SHARED_X-1][SHARED_Y-1];
        }
    }
    // set inner score (aka sequence match/mismatch score):
    char charS1 = s_seq1[tIDx];
    char charS2 = s_seq2[tIDy];

    innerScore = charS1 == FILL_CHARACTER || charS2 == FILL_CHARACTER ? FILL_SCORE : scoringsMatrix[charS1-characterOffset][charS2-characterOffset];

    // transpose the index
    ++tIDx;
    ++tIDy;
    // set shared matrix to zero (starting point!)
    s_matrix[tIDx][tIDy] = 0.0f;
    s_matrix_i[tIDx][tIDy] = AFFINE_GAP_INIT;
    s_matrix_j[tIDx][tIDy] = AFFINE_GAP_INIT;


    // wait until all elements have been copied to the shared memory block
        /**** sync barrier ****/
    __syncthreads();

    currentScore = 0.0f;

    for (int i=0; i < DIAGONAL; ++i) {
        if (i == tXM1+ tYM1) {
        	// calculate only when there are two valid characters
        	// this is necessary when the two sequences are not of equal length
        	// this is the SW-scoring of the cell:

        	m_M = s_matrix[tXM1][tYM1]+innerScore;
        	m_I = s_matrix_i[tXM1][tYM1]+innerScore;
        	m_J = s_matrix_j[tXM1][tYM1]+innerScore;

        	if (currentScore < m_I) { // score comes from I matrix (gap in x)
        		currentScore = m_I;
        		direction = A_DIRECTION | MAIN_MATRIX;
        	}
        	if (currentScore < m_J) { // score comes from J matrix (gap in y)
        		currentScore = m_J;
        		direction = A_DIRECTION | MAIN_MATRIX;
        	}
        	if (currentScore < m_M) { // score comes from m matrix (match)
        		currentScore = m_M;
        		direction = A_DIRECTION | MAIN_MATRIX;
        	}
        	s_matrix[tIDx][tIDy] = innerScore == FILL_SCORE ? 0.0 : currentScore; // copy score to matrix

        	// now do I matrix:
        	currentScore_i = AFFINE_GAP_INIT;
        	m_M = gapScore + gapExtension + s_matrix[tIDx][tYM1];
			m_I = gapExtension + s_matrix_i[tIDx][tYM1];

			if (currentScore_i < m_I) { // score comes from I matrix (gap in x)
        		currentScore_i = m_I;
        		direction_i = B_DIRECTION | I_MATRIX;
        	}
        	if (currentScore_i < m_M) { // score comes from m matrix (match)
        		currentScore_i = m_M;
        		direction_i= B_DIRECTION | I_MATRIX;
        	}
        	s_matrix_i[tIDx][tIDy] = currentScore_i < 0 ? AFFINE_GAP_INIT : currentScore_i; // copy score to matrix

        	// now do J matrix:
        	currentScore_j = AFFINE_GAP_INIT;
        	m_M = gapScore + gapExtension + s_matrix[tXM1][tIDy];
			m_J = gapExtension + s_matrix_j[tXM1][tIDy];

        	if (currentScore_j < m_J) { // score comes from J matrix (gap in y)
        		currentScore_j = m_J;
        		direction_j = C_DIRECTION | J_MATRIX;
        	}
        	if (currentScore_j < m_M) { // score comes from m matrix (match)
        		currentScore_j = m_M;
        		direction_j = C_DIRECTION | J_MATRIX;
        	}
        	s_matrix_j[tIDx][tIDy] = currentScore_j < 0 ? AFFINE_GAP_INIT : currentScore_j; // copy score to matrix

        	currentScore = fmax(currentScore,fmax(currentScore_i,currentScore_j));
        	if (currentScore > 0) {
				if (currentScore == s_matrix[tIDx][tIDy]) {// direction from main
					direction = direction;
				}
				else if(currentScore == s_matrix_i[tIDx][tIDy]) {// direction from I
					direction = direction_i;
				}
				else if(currentScore == s_matrix_j[tIDx][tIDy]){ // direction from J
					direction = direction_j;
				}
        	}
        }

        else if (i-1 == tXM1 + tYM1 ){
                // use this to find max

            if (i==1) {
                s_maxima[0][0] = max(maxPrev, currentScore);
            }
            else if (!tXM1 && tYM1) {
                s_maxima[0][tYM1] = max(s_maxima[0][tYM1-1], currentScore);
            }
            else if (!tYM1 && tXM1) {
                s_maxima[tXM1][0] = max(s_maxima[tXM1-1][0], currentScore);
            }
            else if (tXM1 && tYM1 ){
                s_maxima[tXM1][tYM1] = max(s_maxima[tXM1-1][tYM1], max(s_maxima[tXM1][tYM1-1], currentScore));
            }
        }
        // wait until all threads have calculated their new score
            /**** sync barrier ****/
        __syncthreads();
    }
    // copy end score to the scorings matrix:
    (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx][blocky].value[tXM1][tYM1] = s_matrix[tIDx][tIDy];
    (*matrix_i).metaMatrix[bIDx][bIDy].matrix[blockx][blocky].value[tXM1][tYM1] = s_matrix_i[tIDx][tIDy];
    (*matrix_j).metaMatrix[bIDx][bIDy].matrix[blockx][blocky].value[tXM1][tYM1] = s_matrix_j[tIDx][tIDy];
    (*globalDirection).direction[bIDx][bIDy].localDirection[blockx][blocky].value[tXM1][tYM1] = direction;

    if (tIDx==SHARED_X && tIDy==SHARED_Y)
        globalMaxima->blockMaxima[bIDx][bIDy].value[blockx][blocky] = max(currentScore, max(s_maxima[SHARED_X-2][SHARED_Y-1], s_maxima[SHARED_X-1][SHARED_Y-2]));

    // wait until all threads have copied their score:
        /**** sync barrier ****/
    __syncthreads();
}




__global__ void traceback(GlobalMatrix *matrix, unsigned int x, unsigned int y, unsigned int numberOfBlocks, GlobalMaxima *globalMaxima, GlobalDirection *globalDirection, GlobalDirection *globalDirectionZeroCopy, unsigned int *indexIncrement, StartingPoints *startingPoints, float *maxPossibleScore) {
    /**
     * shared memory block for calculations. It requires
     * extra (+1 in both directions) space to hold
     * Neighboring cells
     */
    __shared__ float s_matrix[SHARED_X+1][SHARED_Y+1];
    /**
     * shared memory for storing the maximum value of this alignment.
     */
    __shared__ float s_maxima[1];
    __shared__ float s_maxPossibleScore[1];

    // calculate indices:
    unsigned int yDIVnumSeq = (blockIdx.y/NUMBER_TARGETS);
    unsigned int blockx = x - yDIVnumSeq;
    unsigned int blocky = y + yDIVnumSeq;
    unsigned int tIDx = threadIdx.x;
    unsigned int tIDy = threadIdx.y;
    unsigned int bIDx = blockIdx.x;
    unsigned int bIDy = blockIdx.y%NUMBER_TARGETS;

    float value;

    if (!tIDx && !tIDy) {
        s_maxima[0] = globalMaxima->blockMaxima[bIDx][bIDy].value[XdivSHARED_X-1][YdivSHARED_Y-1];
        s_maxPossibleScore[0] =  maxPossibleScore[bIDy*NUMBER_SEQUENCES+bIDx];
    }

    __syncthreads();
    if (s_maxima[0]>= MINIMUM_SCORE) { // if the maximum score is below threshold, there is nothing to do

        s_matrix[tIDx][tIDy] = (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx][blocky].value[tIDx][tIDy];

        unsigned char direction = globalDirection->direction[bIDx][bIDy].localDirection[blockx][blocky].value[tIDx][tIDy];


        // wait until all elements have been copied to the shared memory block
        /**** sync barrier ****/
        __syncthreads();

        for (int i=DIAGONAL-1; i >= 0; --i) {

            if ((i == tIDx + tIDy) && direction == UPPER_LEFT_DIRECTION && s_matrix[tIDx][tIDy] >= LOWER_LIMIT_SCORE * s_maxima[0] && s_matrix[tIDx][tIDy] >= s_maxPossibleScore[0]) {
                // found starting point!
                // reserve index:
                unsigned int index = atomicAdd(indexIncrement, 1);
                // now copy this to host:
                StartingPoint *start = &(startingPoints->startingPoint[index]);
                start->sequence = bIDx;
                start->target = bIDy;
                start->blockX = blockx;
                start->blockY = blocky;
                start->valueX = tIDx;
                start->valueY = tIDy;
                start->score = s_matrix[tIDx][tIDy];
                start->maxScore = s_maxima[0];
                start->posScore = s_maxPossibleScore[0];
                //                startingPoints->startingPoint[index] = start;
                // mark this value:
                s_matrix[tIDx][tIDy] = __int_as_float(SIGN_BIT_MASK | __float_as_int(s_matrix[tIDx][tIDy]));
            }
                
            __syncthreads();

            if ((i == tIDx + tIDy) && s_matrix[tIDx][tIDy] < 0 && direction == UPPER_LEFT_DIRECTION) {
                if (tIDx && tIDy){
                    value = s_matrix[tIDx-1][tIDy-1];
                    if (value == 0.0f)
                        direction = STOP_DIRECTION;
                    else
                        s_matrix[tIDx-1][tIDy-1] = __int_as_float(SIGN_BIT_MASK | __float_as_int(value));
                }
                else if (!tIDx && tIDy && blockx) {
                    value = (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky].value[SHARED_X-1][tIDy-1];
                    if (value == 0.0f)
                        direction = STOP_DIRECTION;
                    else
                        (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky].value[SHARED_X-1][tIDy-1] = __int_as_float(SIGN_BIT_MASK | __float_as_int(value));
                }
                else if (!tIDx && !tIDy && blockx && blocky) {
                    value = (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky-1].value[SHARED_X-1][SHARED_Y-1];
                    if (value == 0.0f)
                        direction = STOP_DIRECTION;
                    else
                        (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky-1].value[SHARED_X-1][SHARED_Y-1] = __int_as_float(SIGN_BIT_MASK | __float_as_int(value));
                }
                else if (tIDx && !tIDy && blocky) {
                    value = (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx][blocky-1].value[tIDx-1][SHARED_Y-1];
                    if (value == 0.0f)
                        direction = STOP_DIRECTION;
                    else
                        (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx][blocky-1].value[tIDx-1][SHARED_Y-1] = __int_as_float(SIGN_BIT_MASK | __float_as_int(value));
                }
            }
            __syncthreads();

            if ((i == tIDx + tIDy) && s_matrix[tIDx][tIDy] < 0 && direction == UPPER_DIRECTION) {
                if (!tIDy) {
                    if (blocky) {
                        value = (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx][blocky-1].value[tIDx][SHARED_Y-1];
                        if (value == 0.0f)
                            direction = STOP_DIRECTION;
                        else
                            (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx][blocky-1].value[tIDx][SHARED_Y-1] = __int_as_float(SIGN_BIT_MASK | __float_as_int(value));
                    }
                }
                else {
                    value = s_matrix[tIDx][tIDy-1];
                    if (value == 0.0f)
                        direction = STOP_DIRECTION;
                    else
                        s_matrix[tIDx][tIDy-1] = __int_as_float(SIGN_BIT_MASK | __float_as_int(value));
                }
            }

            __syncthreads();
            if ((i == tIDx + tIDy) && s_matrix[tIDx][tIDy] < 0 && direction == LEFT_DIRECTION) {
                if (!tIDx){
                    if (blockx) {
                        value = (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky].value[SHARED_X-1][tIDy];
                        if (value == 0.0f)
                            direction = STOP_DIRECTION;
                        else
                            (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky].value[SHARED_X-1][tIDy] = __int_as_float(SIGN_BIT_MASK | __float_as_int(value));
                    }
                }
                else {
                    value = s_matrix[tIDx-1][tIDy];
                    if (value == 0.0f)
                        direction = STOP_DIRECTION;
                    else
                        s_matrix[tIDx-1][tIDy] = __int_as_float(SIGN_BIT_MASK | __float_as_int(value));
                }
            }

            __syncthreads();

        }

        // copy end score to the scorings matrix:
        if (s_matrix[tIDx][tIDy] < 0) {
            (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx][blocky].value[tIDx][tIDy] = s_matrix[tIDx][tIDy];
            globalDirectionZeroCopy->direction[bIDx][bIDy].localDirection[blockx][blocky].value[tIDx][tIDy] = direction;
        }
        /**** sync barrier ****/
        __syncthreads();
    }
}

__device__ unsigned char tracebackStepLeftUp(unsigned int blockx, unsigned int blocky, float s_matrix[][SHARED_Y+1], GlobalMatrix *matrix, unsigned char direction){
    unsigned int tIDx = threadIdx.x;
    unsigned int tIDy = threadIdx.y;
    unsigned int bIDx = blockIdx.x;
    unsigned int bIDy = blockIdx.y%NUMBER_TARGETS;
    unsigned char dir = direction;
    float value;

	if (tIDx && tIDy){
		value = s_matrix[tIDx-1][tIDy-1];
		if (value == 0.0f)
			dir = STOP_DIRECTION;
		else
			s_matrix[tIDx-1][tIDy-1] = __int_as_float(SIGN_BIT_MASK | __float_as_int(value));
	}
	else if (!tIDx && tIDy && blockx) {
		value = (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky].value[SHARED_X-1][tIDy-1];
		if (value == 0.0f)
			dir = STOP_DIRECTION;
		else
			(*matrix).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky].value[SHARED_X-1][tIDy-1] = __int_as_float(SIGN_BIT_MASK | __float_as_int(value));
	}
	else if (!tIDx && !tIDy && blockx && blocky) {
		value = (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky-1].value[SHARED_X-1][SHARED_Y-1];
		if (value == 0.0f)
			dir = STOP_DIRECTION;
		else
			(*matrix).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky-1].value[SHARED_X-1][SHARED_Y-1] = __int_as_float(SIGN_BIT_MASK | __float_as_int(value));
	}
	else if (tIDx && !tIDy && blocky) {
		value = (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx][blocky-1].value[tIDx-1][SHARED_Y-1];
		if (value == 0.0f)
			dir = STOP_DIRECTION;
		else
			(*matrix).metaMatrix[bIDx][bIDy].matrix[blockx][blocky-1].value[tIDx-1][SHARED_Y-1] = __int_as_float(SIGN_BIT_MASK | __float_as_int(value));
	}
	return dir;

}

__device__ unsigned char tracebackStepUp(unsigned int blockx, unsigned int blocky, float s_matrix[][SHARED_Y+1], GlobalMatrix *matrix, unsigned char direction){
    unsigned int tIDx = threadIdx.x;
    unsigned int tIDy = threadIdx.y;
    unsigned int bIDx = blockIdx.x;
    unsigned int bIDy = blockIdx.y%NUMBER_TARGETS;
    unsigned char dir = direction;
    float value;

    if (!tIDy) {
        if (blocky) {
            value = (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx][blocky-1].value[tIDx][SHARED_Y-1];
            (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx][blocky-1].value[tIDx][SHARED_Y-1] = __int_as_float(SIGN_BIT_MASK | __float_as_int(value));
        }
    }
    else {
        value = s_matrix[tIDx][tIDy-1];
        s_matrix[tIDx][tIDy-1] = __int_as_float(SIGN_BIT_MASK | __float_as_int(value));
    }
	return dir;

}

__device__ unsigned char tracebackStepLeft(unsigned int blockx, unsigned int blocky, float s_matrix[][SHARED_Y+1], GlobalMatrix *matrix, unsigned char direction){
    unsigned int tIDx = threadIdx.x;
    unsigned int tIDy = threadIdx.y;
    unsigned int bIDx = blockIdx.x;
    unsigned int bIDy = blockIdx.y%NUMBER_TARGETS;
    unsigned char dir = direction;
    float value;

    if (!tIDx){
        if (blockx) {
            value = (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky].value[SHARED_X-1][tIDy];
            (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx-1][blocky].value[SHARED_X-1][tIDy] = __int_as_float(SIGN_BIT_MASK | __float_as_int(value));
        }
    }
    else {
        value = s_matrix[tIDx-1][tIDy];
        s_matrix[tIDx-1][tIDy] = __int_as_float(SIGN_BIT_MASK | __float_as_int(value));
    }


    return dir;

}

__global__ void tracebackAffineGap(GlobalMatrix *matrix, GlobalMatrix *matrix_i, GlobalMatrix *matrix_j,unsigned int x, unsigned int y, unsigned int numberOfBlocks, GlobalMaxima *globalMaxima, GlobalDirection *globalDirection, GlobalDirection *globalDirectionZeroCopy, unsigned int *indexIncrement, StartingPoints *startingPoints, float *maxPossibleScore){
    /**
     * shared memory block for calculations. It requires
     * extra (+1 in both directions) space to hold
     * Neighboring cells
     */
    __shared__ float s_matrix[SHARED_X+1][SHARED_Y+1];
    __shared__ float s_matrix_i[SHARED_X+1][SHARED_Y+1];
    __shared__ float s_matrix_j[SHARED_X+1][SHARED_Y+1];
    /**
     * shared memory for storing the maximum value of this alignment.
     */
    __shared__ float s_maxima[1];
    __shared__ float s_maxPossibleScore[1];

    // calculate indices:
    unsigned int yDIVnumSeq = (blockIdx.y/NUMBER_TARGETS);
    unsigned int blockx = x - yDIVnumSeq;
    unsigned int blocky = y + yDIVnumSeq;
    unsigned int tIDx = threadIdx.x;
    unsigned int tIDy = threadIdx.y;
    unsigned int bIDx = blockIdx.x;
    unsigned int bIDy = blockIdx.y%NUMBER_TARGETS;



    if (!tIDx && !tIDy) {
        s_maxima[0] = globalMaxima->blockMaxima[bIDx][bIDy].value[XdivSHARED_X-1][YdivSHARED_Y-1];
        s_maxPossibleScore[0] =  maxPossibleScore[bIDy*NUMBER_SEQUENCES+bIDx];
    }

    __syncthreads();


    if (s_maxima[0]>= MINIMUM_SCORE) { // if the maximum score is below threshold, there is nothing to do
        unsigned char direction = DIRECTION_MASK & globalDirection->direction[bIDx][bIDy].localDirection[blockx][blocky].value[tIDx][tIDy];
        unsigned char matrix_source = MATRIX_MASK & globalDirection->direction[bIDx][bIDy].localDirection[blockx][blocky].value[tIDx][tIDy];

    	s_matrix[tIDx][tIDy] = (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx][blocky].value[tIDx][tIDy];
        s_matrix_i[tIDx][tIDy] = (*matrix_i).metaMatrix[bIDx][bIDy].matrix[blockx][blocky].value[tIDx][tIDy];
        s_matrix_j[tIDx][tIDy] = (*matrix_j).metaMatrix[bIDx][bIDy].matrix[blockx][blocky].value[tIDx][tIDy];

        // wait until all elements have been copied to the shared memory block
        /**** sync barrier ****/
        __syncthreads();

        for (int i=DIAGONAL-1; i >= 0; --i) {

            if ((i == tIDx + tIDy) && matrix_source == MAIN_MATRIX && s_matrix[tIDx][tIDy] >= LOWER_LIMIT_SCORE * s_maxima[0] && s_matrix[tIDx][tIDy] >= s_maxPossibleScore[0]) {
                // found starting point!
                // reserve index:
                unsigned int index = atomicAdd(indexIncrement, 1);
                // now copy this to host:
                StartingPoint *start = &(startingPoints->startingPoint[index]);
                start->sequence = bIDx;
                start->target = bIDy;
                start->blockX = blockx;
                start->blockY = blocky;
                start->valueX = tIDx;
                start->valueY = tIDy;
                start->score = s_matrix[tIDx][tIDy];
                start->maxScore = s_maxima[0];
                start->posScore = s_maxPossibleScore[0];
                //                startingPoints->startingPoint[index] = start;
                // mark this value:
                s_matrix[tIDx][tIDy] = __int_as_float(SIGN_BIT_MASK | __float_as_int(s_matrix[tIDx][tIDy]));
            }

            __syncthreads();

            if ((i == tIDx + tIDy) && (
            		(s_matrix[tIDx][tIDy] < 0 && matrix_source == MAIN_MATRIX) ||
            		(s_matrix_i[tIDx][tIDy] < 0 && s_matrix_i[tIDx][tIDy] > AFFINE_GAP_INIT && matrix_source == I_MATRIX) ||
            		(s_matrix_j[tIDx][tIDy] < 0 && s_matrix_j[tIDx][tIDy] > AFFINE_GAP_INIT && matrix_source == J_MATRIX)
					)) {
					// check which matrix to go to:
					switch (direction) {
					case A_DIRECTION : // M
						direction = tracebackStepLeftUp(blockx, blocky, s_matrix, matrix, direction);
						break;
					case B_DIRECTION : // I
						direction = tracebackStepUp(blockx, blocky, s_matrix_i, matrix_i, direction);
						break;
					case C_DIRECTION : // J
						direction = tracebackStepLeft(blockx, blocky, s_matrix_j, matrix_j, direction);
						break;
						}
			}
            __syncthreads();

        }

        if (matrix_source == MAIN_MATRIX) {
            (*matrix).metaMatrix[bIDx][bIDy].matrix[blockx][blocky].value[tIDx][tIDy] = s_matrix[tIDx][tIDy];
            globalDirectionZeroCopy->direction[bIDx][bIDy].localDirection[blockx][blocky].value[tIDx][tIDy] = direction;
        }
        else if (matrix_source == I_MATRIX) {
            (*matrix_i).metaMatrix[bIDx][bIDy].matrix[blockx][blocky].value[tIDx][tIDy] = s_matrix_i[tIDx][tIDy];
            globalDirectionZeroCopy->direction[bIDx][bIDy].localDirection[blockx][blocky].value[tIDx][tIDy] = direction;
        }
        else if (matrix_source == J_MATRIX) {
            (*matrix_j).metaMatrix[bIDx][bIDy].matrix[blockx][blocky].value[tIDx][tIDy] = s_matrix_j[tIDx][tIDy];
            globalDirectionZeroCopy->direction[bIDx][bIDy].localDirection[blockx][blocky].value[tIDx][tIDy] = direction;
        }
        /**** sync barrier ****/
        __syncthreads();
    }
}
