
#include <hip/hip_runtime.h>
/** Direction definitions for the direction matrix. These are needed for the trace back */
#define NO_DIRECTION ${NO_DIRECTION}
#define STOP_DIRECTION ${STOP_DIRECTION}

#define UPPER_LEFT_DIRECTION ${UP_LEFT_DIRECTION}
#define UPPER_DIRECTION ${UP_DIRECTION}
#define LEFT_DIRECTION ${LEFT_DIRECTION}

/** Direction definitions for affine gap penalty **/
#define MAIN_MATRIX (${UP_LEFT_DIRECTION}  << 2)
#define I_MATRIX (${UP_DIRECTION} << 2)
#define J_MATRIX (${LEFT_DIRECTION} << 2)

#define A_DIRECTION ${UP_LEFT_DIRECTION}
#define B_DIRECTION ${UP_DIRECTION}
#define C_DIRECTION ${LEFT_DIRECTION}
#define DIRECTION_MASK 3
#define MATRIX_MASK 12

