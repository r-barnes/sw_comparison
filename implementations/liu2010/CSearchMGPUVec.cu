#include "hip/hip_runtime.h"
/***********************************************
* # Copyright 2009. Liu Yongchao
* # Contact: Liu Yongchao
* #          liuy0039@ntu.edu.sg; nkcslyc@hotmail.com
* #
* # GPL 2.0 applies.
* #
* ************************************************/

#include "CSearchMGPUVec.h"
#include "CFastaSWVec.h"
#include "CFastaFile.h"
#include <pthread.h>

CSearchMGPUVec::CSearchMGPUVec(CParams* params) : CSearchVec(params)
{
	//initialize the global host result buffer address to zero
	globalHostResult = 0;
}
CSearchMGPUVec::~CSearchMGPUVec()
{
	if(globalHostResult){
		pFreeHost(globalHostResult);
	}
}

int CSearchMGPUVec::dbsearch(char* queryFile)
{
	int i,j;
    CFastaFile* queryLib = new CFastaFile;

	//build coalesced sequence array for the sorted database sequence indexed less than numThreshold
	int n;
	int threads;
	int slen;
	unsigned char* seq;
	int width, height;

	//compute the width of the array
	//must make sure threads = 256; if 256 threads in a thread block is not allowed due to
	//the number of available registers, please modify the loading substitution matrix CUDA code
	threads = THREADS_PER_BLOCK;
	//malloc global host result buffer
	globalHostResult = (SeqEntry*)pMallocHost(sizeof(SeqEntry) * numSeqs);
#define MINIMAL_WIDTH           (THREADS_PER_BLOCK * 30)
	//initialize the TaskPlan objects
	GPUInfo* info = pGetGPUInfo();
	TaskPlan * plans = (TaskPlan*) malloc (info->n_device * sizeof(TaskPlan));
	for(i = 0; i < info->n_device; i++){
		plans[i].threads = threads;
		plans[i].device = i;
		plans[i].info = info;
		plans[i].cudaInterTexWidth =  max(pGetMultiProcessorCount(info, i) * threads, MINIMAL_WIDTH);
		plans[i].cudaInterTexHeight = 0;
		plans[i].maxSeqLength = maxSeqLength;
		plans[i].cx = plans[i].cy = 0;
		//
		plans[i].hostSeqHash = (DatabaseHash*)pMallocHost(numSeqs * sizeof(DatabaseHash));
		plans[i].numSeqs = 0;
		plans[i].interSeqNo = 0;
		plans[i].intraSeqNo = 0;
		plans[i].cudasw = new CFastaSWVec;
	}

	/*****************************************************************
	***************(Inter-Task Parallelization) Stage 1***************
 	*****************************************************************/

	//allocate sequences for each GPU
	bool done;
	do{
		i = 0;
		done = true;
		while(i < numThreshold){
			for(int dev = 0; dev < info->n_device; dev++){
				TaskPlan* plan = &plans[dev];
				if(i >= numThreshold){
					break;
				}
				int idx = sortedSeqs[i].idx;
				n = dbSeqsLen[idx];
				if(n + plan->cx > plan->cudaInterTexWidth){
					plan->cx = 0;
					plan->cy ++;
				}
				plan->cx += n;

				//increase the number of sequence
				plan->interSeqNo ++;
				//increase the total number of sequence
				i++;
			}
		}
		//compute the real height
		for(i = 0; i < info->n_device; i++){
			plans[i].cudaInterTexHeight = plans[i].cy + 1;
			if(plans[i].cudaInterTexHeight > 32768){
				for(int j = 0; j < info->n_device; j++){
					plans[j].cudaInterTexHeight = 0;
					plans[j].cudaInterTexWidth = 65536;
					plans[j].interSeqNo = 0;
					plans[j].cx = 0;
					plans[j].cy = 0;
				}
				done = false;
				break;
			}
		}
	}while(!done);

	for(i = 0; i < info->n_device; i++){
		width = plans[i].cudaInterTexWidth;
		height = plans[i].cudaInterTexHeight;
		if(width > 65536 || height > 32768){
			fprintf(stderr, "width(%d) or height(%d) out of texture reference range\n", width, height);
			return 1;
		}
	}

	//fill the array with the sorted sequences
	for(i = 0; i < info->n_device; i++){
		plans[i].cx = 0;
		plans[i].cy = 0;
		if(plans[i].cudaInterTexHeight == 0){
			plans[i].cudaInterTexHeight = 1;
		}
		plans[i].interHostSeqArray = pMallocHost(
				plans[i].cudaInterTexWidth * plans[i].cudaInterTexHeight * sizeof(unsigned char));

		//allocate result slot for host
		plans[i].hostResult = (SeqEntry*)pMallocHost(sizeof(SeqEntry) * numSeqs);
		plans[i].hostResultPos = 0;
		plans[i].globalHostResult = globalHostResult;
	}

	i = 0;
	while(i < numThreshold){
		for( int dev = 0; dev < info->n_device; dev++){
			if( i >= numThreshold){
				break;
			}
			TaskPlan* plan = &plans[dev];
			width = plan->cudaInterTexWidth;
			height = plan->cudaInterTexHeight;
			unsigned char* array = (unsigned char*)plan->interHostSeqArray;

			int idx = sortedSeqs[i].idx;
			seq = dbSeqs[idx];
			n = dbSeqsLen[idx];
        	if(n + plan->cx > width){
            	//adjust the coordinates
            	plan->cx = 0;
            	plan->cy ++;
        	}
        	//copy the sequence
        	unsigned char* ptr = array + plan->cy * width + plan->cx;
        	memcpy(ptr, seq, n * sizeof(unsigned char));

			//build the corresponding hash item
			int index = plan->numSeqs;
			plan->hostSeqHash[index].cx = plan->cx;
			plan->hostSeqHash[index].cy = plan->cy;
			plan->hostSeqHash[index].length = n;
			plan->hostSeqHash[index].alignedLen = dbSeqsAlignedLen[idx];

			//save the corresponding sequence index and initialize the value
			plan->hostResult[index].idx = idx;
			plan->hostResult[index].value = 65536;
			plan->numSeqs ++;

			//adjust the coordinates
			plan->cx += n;

			if(plan->cy >= height){
				printf("the array overflowed at the bottom (cy:%d heigth:%d)! press any key to continue\n",
							plan->cy, height);
				getchar();
				break;
			}

			//increase the total number of sequence
			i++;
		}
	}
    /*****************************************************************
 	***************Intra-Task Parallelization Stage 2*****************
 	******************************************************************/

	if(numThreshold == numSeqs){
		//pseudo number to avoid binding errors
		for(i = 0; i < info->n_device; i++){
		 	plans[i].cx = 0;
        	plans[i].cy = 0;
       	 	plans[i].cudaIntraTexWidth = 1;  //not out of the texture reference range
        	//set maximum height
        	plans[i].cudaIntraTexHeight = 1;

        	//allocate host memory
        	plans[i].intraHostSeqArray = (unsigned char*) pMallocHost(plans[i].cudaIntraTexWidth *
                	plans[i].cudaIntraTexHeight * sizeof(unsigned char));
		}
		goto stage3;
	}
	for(i = 0; i < info->n_device; i++){
		plans[i].cx = 0;
		plans[i].cy = 0;
		plans[i].cudaIntraTexWidth = maxSeqLength + 1;	//not out of the texture reference range
		//set maximum height
		plans[i].cudaIntraTexHeight = (numSeqs - numThreshold + info->n_device - 1)/info->n_device;

		//allocate host memory
    if(plans[i].cudaIntraTexHeight <= 0){
      plans[i].cudaIntraTexHeight = 0;
    }
		plans[i].intraHostSeqArray = (unsigned char*) pMallocHost(plans[i].cudaIntraTexWidth *
				plans[i].cudaIntraTexHeight * sizeof(unsigned char));
	}

	n = numThreshold;
	while ( n < numSeqs){
		for(i = 0; i < info->n_device; i++){

			width = plans[i].cudaIntraTexWidth;
			if(n >= numSeqs){
				break;
			}
			//get the sequence and its length
			int idx = sortedSeqs[n].idx;
			seq = dbSeqs[idx];
			slen = dbSeqsLen[idx];

			if(slen + 1 + plans[i].cx > width){
				//adjust the coordinates
				plans[i].cx = 0;
				plans[i].cy ++;
			}
			//copy the sequence
			unsigned char* ptr = plans[i].intraHostSeqArray + plans[i].cy * width + plans[i].cx;
			for(j = 0; j < slen; j++){
				ptr[j + 1] = seq[j];
			}
 			//build the corresponding hash item
         	int index = plans[i].numSeqs;
         	plans[i].hostSeqHash[index].cx = plans[i].cx;
          	plans[i].hostSeqHash[index].cy = plans[i].cy;
       		plans[i].hostSeqHash[index].length = slen;

       		//save the corresponding sequence index and initialize the value
         	plans[i].hostResult[index].idx = idx;
         	plans[i].hostResult[index].value = 65536;
           	plans[i].numSeqs ++;
			plans[i].intraSeqNo++;

			//adjust the coordinates
			plans[i].cx += slen + 1;

			//increase n
			n++;
		}
	}

	for(i = 0; i < info->n_device; i++){
		plans[i].cudaIntraTexHeight = plans[i].cy + 1;
	}
   	/*****************************************************************
 	*********************************Stage 3***************************
 	******************************************************************/
stage3:

	int pos = 0;
	for(i = 0; i < info->n_device; i++){
		if(plans[i].interSeqNo + plans[i].intraSeqNo != plans[i].numSeqs){
			printf("seq number error-----i:%d\n",i);
			getchar();
			return 0;
		}
		//initialize the results buffer on the GPU
		plans[i].hostResultPos = pos;
		pos += plans[i].numSeqs;
	}
	printf("Loading database successfully\n");

	printf("numSeqs: %d numThreshold: %d\n", numSeqs, numThreshold);
	printf("maxSeqLength: %d totalAminoAcids: %d\n", maxSeqLength, totalAminoAcids);

	printf("******************************\n");
	printf("******************************\n");
	//load queries
	int qlen;
	int qAlignedLen;
	unsigned char* query;

	/*thread ID*/
 	pthread_t * threadID = (pthread_t*)malloc(sizeof(pthread_t) * info->n_device);

	//open the query file
    queryLib->open(queryFile);
	//only load the first query sequence
	query = queryLib->nextSeq(&qlen, &qAlignedLen, SEQ_LENGTH_ALIGNED);
	if(qlen == 0){
		printf("query file is empty!");
		goto out;
	}
	while(qlen > 0){
		double start, end;
		//get the systeme time
		CParams::getSysTime(&start);

		for(i = 0; i < info->n_device; i++){
			plans[i].qLen = qlen;
			plans[i].qAlignedLen = qAlignedLen;
			plans[i].query = query;
		}
		//create threads
		for(i = 0; i < info->n_device;i++){
			/*create a thread*/
			pthread_create(&threadID[i], NULL, swthreads_func, (void*)&plans[i]);
		}
		/*wait for the completion of threads*/
		for(i = 0; i < info->n_device; ++i){
			pthread_join(threadID[i], NULL);
		}

		//get the system time
		CParams::getSysTime(&end);

		double dif = end - start;
		double gcups = ((float)totalAminoAcids)/1000000.0;
		gcups /= 1000.0;
		gcups *= qlen;
		gcups /= dif;

		#ifndef BENCHMARKING
			printf("query:%s \n", queryLib->getSeqName());
			printf("Length: %d --- time: %g (s) GCUPS: %g\n", qlen, dif, gcups);
		#endif

		//display results
		int top = numSeqs > params->getTopScoresNum() ? params->getTopScoresNum(): numSeqs;
        int scoreThreshold = params->getScoreThreshold();
		printResults(globalHostResult, dbSeqsName, numSeqs, top, scoreThreshold);
		//load the next query sequence
		query = queryLib->nextSeq(&qlen, &qAlignedLen, SEQ_LENGTH_ALIGNED);
    	if(qlen == 0){
        	printf("Reaching the end of the query file!\n");
    	}
	}

out:
	for(i = 0; i < info->n_device; i++){
		TaskPlan * plan = &plans[i];
		pFreeHost(plan->interHostSeqArray);
		pFreeHost(plan->intraHostSeqArray);
		pFreeHost(plan->hostSeqHash);
		pFreeHost(plan->hostResult);
		delete plan->cudasw;
	}
	free(plans);
	free(info);

	if(globalHostResult){
		pFreeHost(globalHostResult);
		globalHostResult = 0;
	}

	//close the database files
	delete queryLib;

    return 0;
}
void* CSearchMGPUVec::swthreads_func(void *arg)
{
	int n;
	int threads;
	int blocks;
	int blk;
	int width, height;
	TaskPlan* plan = (TaskPlan*)arg;
	CFastaSW* cudasw = plan->cudasw;

	//select device
	pSetDevice(plan->info, plan->info->devices[plan->device]);

 	//copy the subsitution matrix, gap penalties from host to GPU
   	cudasw->swMemcpyParameters(CSearch::matrix, CSearch::gapOpen, CSearch::gapExtend);

	//copy the inter-task database sequences from host to GPU
	width = plan->cudaInterTexWidth;
	height = plan->cudaInterTexHeight;

	cudasw->cudaInterSeqs = cudasw->swMallocArray(width, height, pChannelFormatKindUnsignedChar);
	pMemcpyToArray(cudasw->cudaInterSeqs, 0, 0, plan->interHostSeqArray,
                width * height * sizeof(unsigned char), pMemcpyHostToDevice);

	//copy the intra-task sequences into hipArray
    width = plan->cudaIntraTexWidth;
 	height = plan->cudaIntraTexHeight;
	cudasw->cudaIntraSeqs = cudasw->swMallocArray(width, height, pChannelFormatKindUnsignedChar);
	pMemcpyToArray(cudasw->cudaIntraSeqs, 0, 0, plan->intraHostSeqArray,
					width * height * sizeof(unsigned char), pMemcpyHostToDevice);

  	//bind the CUDA Array to texture
	cudasw->swBindTextureToArray();

 	//copy the hash table from host to GPU
	cudasw->cudaSeqHash = (DatabaseHash*)pMallocPitch(sizeof(DatabaseHash),
                plan->numSeqs, 1, 0);
	pMemcpy(cudasw->cudaSeqHash, plan->hostSeqHash,
                plan->numSeqs * sizeof(DatabaseHash), pMemcpyHostToDevice);

	//initialize the results buffer on the GPU
	memcpy(plan->globalHostResult +  plan->hostResultPos, plan->hostResult, plan->numSeqs * sizeof(SeqEntry));
	cudasw->hostResult = &plan->globalHostResult[plan->hostResultPos];

	cudasw->cudaResult = (SeqEntry*)pMallocPitch(sizeof(SeqEntry), plan->numSeqs, 1, 0);
	pMemcpy(cudasw->cudaResult, cudasw->hostResult, plan->numSeqs * sizeof(SeqEntry), pMemcpyHostToDevice);

    //copy the query sequence
    cudasw->swMemcpyQuery(plan->query, plan->qLen, plan->qAlignedLen, sizeof(unsigned char), CSearch::matrix);

	//compute the total number of thread blocks
	threads = plan->threads;
	int warpNum = threads / THREADS_PER_WARP;
	blocks = (plan->interSeqNo + warpNum - 1) / warpNum;

	//allocate memory slots for intermediate results
	int memSlotSize;
	memSlotSize = (SEQ_LENGTH_THRESHOLD + SEQ_LENGTH_ALIGNED - 1) / SEQ_LENGTH_ALIGNED;
	memSlotSize = memSlotSize * SEQ_LENGTH_ALIGNED + 1;

	//allocate memory slot
	int maxProcessors = pGetMultiProcessorCount(plan->info, plan->device);
	int procsPerPass = maxProcessors * 128;
	cudasw->swInterMallocThreadSlots(warpNum, procsPerPass, memSlotSize);
	//binding the query profile
	cudasw->swBindQueryProfile();
	//
	blk = 0;
	while(blocks > 0){
		if(blocks > procsPerPass){
			n = procsPerPass;
		}else{
			n = blocks;
		}
		cudasw->InterRunGlobalDatabaseScanning (n, threads, plan->interSeqNo, blk);
		blk += n;
		blocks -=n;
	}
	//release the memory slots for intermediate results
	cudasw->swInterFreeThreadSlots();
	cudasw->swUnbindQueryProfile();

	if(plan->intraSeqNo > 0){
		//the index of the first sequence in the result buffer
		blk = plan->interSeqNo;
		blocks = plan->intraSeqNo;
		//change the number of threads
		threads = 256;
		//set query sequence length
		cudasw->swMemcpyQueryLength(plan->qLen, plan->qAlignedLen);
		//please decrease the maxSeqsOnePass value
		//when there is no enough global memory on the device
		int maxSeqsOnePass = 256;
		cudasw->swIntraMallocThreadSlots(maxSeqsOnePass, plan->maxSeqLength + 2);
    	while(blocks>0){

			if(blocks > maxSeqsOnePass){
           		n =	maxSeqsOnePass;
        	}else{
           		n = blocks;
       		}
        	cudasw->IntraRunGlobalDatabaseScanning(n, threads, plan->numSeqs, blk);

        	blk += n;
        	blocks -=n;
    	}
		cudasw->swIntraFreeThreadSlots();
	}

	//transfer result from GPU to host
	cudasw->transferResult(plan->numSeqs);

	//free device resources
	cudasw->swUnbindTexture();
	pFree(cudasw->cudaSeqHash);
	pFreeArray(cudasw->cudaInterSeqs);
	pFreeArray(cudasw->cudaIntraSeqs);
	pFree(cudasw->cudaResult);

	return NULL;
}
