#include "hip/hip_runtime.h"
/*
 * Copyright 2012, St. Jude Children's Research Hospital.
 * Written by Pankaj Gupta, pankaj.gupta@stjude.org.
 *
 * This file is part of Swift.  Swift is free software:  you can redistribute
 * it and/or modify it under the terms of the GNU General Public License as
 * published by the Free Software Foundation, either version 2 of the License,
 * or (at your option) any later version.
 *
 * Swift is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
 * A PARTICULAR PURPOSE.  See the GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with Swift.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "common.h"
#include "lookupTable6.h"
#include "preprocess.h"
#include "mapHits6.h"
#include <stdio.h>
#include <limits.h>
#include <time.h>


#define	REF_IDX_BITS2	56
#define	SHIFT_BITS2		28
#define	REF_IDX_MASK2	18374686479671623680 /* Binary: 1111 1111 0000 0000 0000
0000 0000 0000 0000 0000 0000 0000 0000 0000 0000 0000 */
#define	REF_POS_BITS2	25
#define	REF_POS_MASK2	33554431 /* Binary: 1111111111111111111111111 */
#define	REF_POS_MASK3	9151314442816847872 /* Binary: 0111 1111 0000 0000 0000
0000 0000 0000 0000 0000 0000 0000 0000 0000 0000 0000 */
#define	REF_POS_MASK4	268435455 /* Binary: 1111 1111 1111 1111 1111 1111 1111 */


static int *_lookupTable = NULL;
static char *_refIdx = NULL;
static int *_refPos = NULL;
static int *_refPos2 = NULL;
static int _seedLen = 0;
static int _maxHitsPerQry = 0;
static int _numDistinctTuples = 0;
static int *_numRepeatsPerTuple = NULL;
static char *_seed = NULL;
static int _tupleIgnoreThreshold = 0;
static int _maxRefTuplesPerQry = 0;
static int _numTotalTuples = 0;
static int *_numActualRepeatsPerTuple = NULL;
__constant__ int hashCodes_gpu[NUM_ASCII_CHARS];
__constant__ int powerVals_gpu[MAX_SEED_LENGTH];
extern __shared__ int arr_shr[];
__shared__ uint randNumSeed;


/**
 * Creates a lookup table.
 *
 * @param		refFile					Reference file.
 * @param		seedLen					Seed length.
 * @param 		maxHitsPerQry			Maximum hits per query.
 * @param		tupleIgnoreThreshold	Tuple that have number of repeats higher
 * than this value will be ignored.
 * @param[out]	totalTuples				Total number of reference tuples that
 * will be used.
 */
void lookupTable6Create2(const char *refFile, int seedLen, int maxHitsPerQry,
		int tupleIgnoreThreshold, int *totalTuples)
{
	_seedLen = seedLen;
	_maxHitsPerQry = maxHitsPerQry;
	_tupleIgnoreThreshold = tupleIgnoreThreshold;
	_numDistinctTuples = (int) pow((float) DNA_ALPHABET_SIZE, (int) _seedLen);
	_lookupTable = (int *) calloc(_numDistinctTuples, sizeof(int));
	_numActualRepeatsPerTuple = (int *) calloc(_numDistinctTuples, sizeof(int));
	int *numRepeatsPerTuple = (int *) calloc(_numDistinctTuples, sizeof(int));
	_numRepeatsPerTuple = (int *) calloc(_numDistinctTuples, sizeof(int));
	_seed = (char *) calloc((_seedLen + 1), sizeof(char));

	/* First pass through reference to find the number of repeats for each
	 * distinct tuple. */
	time_t startTime, endTime;
	double diffTime;
	time(&startTime);
	fprintf(stderr, "   First pass...");
	FILE *filePtr = fopen(refFile, "r");
	int numIterations, i, j, offset = 0;
	int hashVal, lineLength, numBases = 0, refIdx = -1;
	char line[MAX_LINE_LENGTH];

	char tmpRefFile[MAX_FILE_NAME_LENGTH];
	sprintf(tmpRefFile, "%s%s%s", TMP_DIR, PATH_SEPARATOR, TEMP_REF_FILE);
	FILE *tmpRefFilePtr = fopen(tmpRefFile, "w+");
	_numTotalTuples = 0;
	while (fgets(line + offset, MAX_LINE_LENGTH, filePtr) != NULL)
	{
		lineLength = strlen(line);
		if (line[lineLength - 1] == '\n')
		{
			line[lineLength - 1] = '\0';
			--lineLength;
		}

		/* This is an empty line. */
		if (lineLength == 0)
			offset = 0;
		/* This line contains reference ID. */
		else if (line[offset] == '>')
		{
			offset = 0;
			numBases = 0;
			++refIdx;
		}
		/* This is a line containing sequence. */
		else
		{
			numIterations = lineLength - _seedLen + 1;
			/* Consider non-overlapping tuples only. */
			for (i = 0; i < numIterations; i += _seedLen)
			{
				hashVal = getHash(line + i, _seedLen);
				++_numRepeatsPerTuple[hashVal];
				if (_numRepeatsPerTuple[hashVal] <= _tupleIgnoreThreshold)
				{
					++_numActualRepeatsPerTuple[hashVal];
					++numRepeatsPerTuple[hashVal];
					++_numTotalTuples;
					fprintf(tmpRefFilePtr, "%d\t%d\t%d\n", hashVal, refIdx,
							numBases + i);
				}
				else if (_numRepeatsPerTuple[hashVal]
				                            == _numActualRepeatsPerTuple[hashVal] + 1)
				{
					_numActualRepeatsPerTuple[hashVal] = 0;
					numRepeatsPerTuple[hashVal] = 0;
					_numTotalTuples -= _tupleIgnoreThreshold;
				}
			}
			numBases += lineLength;

			/* Copy the last few bases to the beginning of 'line' array */
			offset = lineLength - i;
			for (j = 0; j < offset; ++j)
				line[j] = line[i + j];
			numBases -= offset;
		}
	}
	*totalTuples = _numTotalTuples;
	_refPos = (int *) calloc((_numTotalTuples + 1), sizeof(int));
	_refPos[0] = -1; /* First element set to -1 so that
	tuples that do not exist in the reference or that have number of repeats
	greater than a threshold value can point to this element. */
	_refPos2 = (int *) calloc((_numTotalTuples + 1), sizeof(int));
	_refPos2[0] = -1;
	_refIdx = (char *) calloc((_numTotalTuples + 1), sizeof(int));
	_refIdx[0] = -1;
	time(&endTime);
	diffTime = difftime(endTime, startTime);
	fprintf(stderr, "done. (Time = %.2lf secs)\n", diffTime);

	/* Set values in the lookup table. */
	time(&startTime);
	fprintf(stderr, "   Set values in lookup table...");
	int start = 1;
	for (i = 0; i < _numDistinctTuples; ++i)
	{
		if (_numRepeatsPerTuple[i] > 0
				&& _numRepeatsPerTuple[i] <= tupleIgnoreThreshold)
		{
			_lookupTable[i] = start;
			start += _numRepeatsPerTuple[i];
		}
	}
	time(&endTime);
	diffTime = difftime(endTime, startTime);
	fprintf(stderr, "done. (Time = %.2lf secs)\n", diffTime);

	/* Set reference offsets. */
	time(&startTime);
	fprintf(stderr, "   Setting reference positions...");
	rewind(tmpRefFilePtr);
	int index, refPos, tmp;
	while (fgets(line, MAX_LINE_LENGTH, tmpRefFilePtr) != NULL)
	{
		sscanf(line, "%d\t%d\t%d", &hashVal, &refIdx, &refPos);
		if (_numRepeatsPerTuple[hashVal] <= _tupleIgnoreThreshold)
		{
			index = _lookupTable[hashVal] + numRepeatsPerTuple[hashVal] - 1;
			_refIdx[index] = (char) refIdx;
			_refPos[index] = refPos;
			tmp = refIdx;
			tmp = tmp << REF_POS_BITS2;
			tmp += (refPos / _seedLen);
			_refPos2[index] = tmp;
			--numRepeatsPerTuple[hashVal];
		}
	}
	time(&endTime);
	diffTime = difftime(endTime, startTime);
	fprintf(stderr, "done. (Time = %.2lf secs)\n", diffTime);

	fprintf(stderr, "   Freeing resources...");
	fclose(tmpRefFilePtr);
	remove(tmpRefFile);
	fclose(filePtr);
	free(numRepeatsPerTuple);
	fprintf(stderr, "done.\n");

	int numQryTuples = MAX_QRY_SEQ_LENGTH - seedLen;
	_maxRefTuplesPerQry = numQryTuples * TUPLE_IGNORE_THRES;
}


/**
 * Reset key variables in this file.
 */
void lookupTable6Reset()
{

}


/**
 * Releases memory occupied by data structures in this file.
 */
void lookupTable6Delete()
{
	free(_lookupTable);
	_lookupTable = NULL;
	free(_refPos);
	_refPos = NULL;
	free(_refPos2);
	_refPos2 = NULL;
	free(_refIdx);
	_refIdx = NULL;
	_seedLen = 0;
	_maxHitsPerQry = 0;
	_numDistinctTuples = 0;
	free(_seed);
	_seed = NULL;
	free(_numRepeatsPerTuple);
	_numRepeatsPerTuple = NULL;
	free(_numActualRepeatsPerTuple);
	_numActualRepeatsPerTuple = NULL;
}


/**
 * Copies hash table from CPU to GPU.
 *
 * @param[out]	keys		Keys array on the GPU.
 * @param[out]	numKeys		Number of elements in @a keys array.
 * @param[out]	values		Values array on the GPU.
 * @param[out]	numValues	Number of elements in @a values array.
 * @param[out]	numRepeatsPerTuple	Number of tuples per hash.
 */
void lookupTable6CpyHashTableToGPU(int **keys, int *numKeys, int **values,
		int *numValues, int **numRepeatsPerTuple)
{
	hipMalloc((void **) keys, _numDistinctTuples * sizeof(int));
	PRINT_CUDA_ERROR()
	hipMemcpy(*keys, _lookupTable, _numDistinctTuples * sizeof(int),
			hipMemcpyHostToDevice);
	PRINT_CUDA_ERROR()
	*numKeys = _numDistinctTuples;

	hipMalloc((void **) values, (_numTotalTuples + 1) * sizeof(int));
	PRINT_CUDA_ERROR()
	hipMemcpy(*values, _refPos2, (_numTotalTuples + 1) * sizeof(int),
			hipMemcpyHostToDevice);
	PRINT_CUDA_ERROR()
	*numValues = _numTotalTuples + 1;

	hipMalloc((void **) numRepeatsPerTuple, _numDistinctTuples * sizeof(int));
	PRINT_CUDA_ERROR()
	hipMemcpy(*numRepeatsPerTuple, _numActualRepeatsPerTuple,
			_numDistinctTuples * sizeof(int), hipMemcpyHostToDevice);
	PRINT_CUDA_ERROR()
}


/**
 * Searches the query sequence in the reference and returns the best-matching
 * reference coordinates.
 *
 * @param	query 				Query object.
 * @param 	seqLength 			Sequence length.
 * @param	refIdx_bestHits		Best-matching reference indexes.
 * @param	shift_bestHits		Best-matching shifts.
 * @param	refPos_bestHits		Best-matching reference positions.
 * @return 	Number of hits in @a refIdx.
 */
int lookupTable6MapQry2(char *qrySeq, int qryLen, char *refIdx_bestHits,
		int *shift_bestHits, int *refPos_bestHits)
{
	int numQryTuples, i, j, k = 0, hash, startIdx, endIdx, numHits;
	char *refIdx = (char *) calloc(_maxRefTuplesPerQry, sizeof(char));
	int *shift = (int *) calloc(_maxRefTuplesPerQry, sizeof(int));
	int *refPos = (int *) calloc(_maxRefTuplesPerQry, sizeof(int));

	/* Step 1: Break the query into tuples. */
	numQryTuples = qryLen - _seedLen + 1;
	for (i = 0; i < numQryTuples; ++i)
	{
		/* Step 2: Calculate the hash for each tuple. */
		/* Step 3: Find the reference coordinates of the hash in the hash
		 * table. */
		/* Step 4: Aggregate the reference coordinate positions. */
		hash = getHash(qrySeq + i, _seedLen);
		if (_numRepeatsPerTuple[hash] > _tupleIgnoreThreshold)
			continue;

		/* Add the tuples to the list. */
		startIdx = _lookupTable[hash];
		endIdx = startIdx + _numRepeatsPerTuple[hash] - 1;
		for (j = startIdx; j <= endIdx; ++j)
		{
			refIdx[k] = _refIdx[j];
			shift[k] = _refPos[j] - i;
			refPos[k] = _refPos[j];
			++k;
		}
	}
	numHits = mapHits6GetBestHits(_maxHitsPerQry, refIdx, shift, refPos, k,
			refIdx_bestHits, shift_bestHits, refPos_bestHits);

	free(refIdx);
	free(shift);
	free(refPos);

	return numHits;
}


/**
 * Copies constant memory from CPU to GPU.
 */
void lookupTable6CpyConstMemToGPU()
{
	int hashCodes[NUM_ASCII_CHARS];
	int i;
	for (i = 0; i < NUM_ASCII_CHARS; ++i)
		hashCodes[i] = 0;
	hashCodes['A'] = CODE_A;
	hashCodes['a'] = CODE_A;
	hashCodes['C'] = CODE_C;
	hashCodes['c'] = CODE_C;
	hashCodes['G'] = CODE_G;
	hashCodes['g'] = CODE_G;
	hashCodes['T'] = CODE_T;
	hashCodes['t'] = CODE_T;
	hashCodes['N'] = CODE_N;
	hashCodes['n'] = CODE_N;
	hipMemcpyToSymbol(HIP_SYMBOL(hashCodes_gpu), hashCodes, sizeof(hashCodes), 0,
			hipMemcpyHostToDevice);
	PRINT_CUDA_ERROR()

	int powerVals[MAX_SEED_LENGTH];
	for (i = 0; i < MAX_SEED_LENGTH; ++i)
		powerVals[i] = (int) pow((float) DNA_ALPHABET_SIZE, i);
	hipMemcpyToSymbol(HIP_SYMBOL(powerVals_gpu), powerVals, sizeof(powerVals), 0,
			hipMemcpyHostToDevice);
	PRINT_CUDA_ERROR()
}


/**
 * Searches the query sequence in the reference on the GPU.
 *
 * @param		keys	Keys of the lookup table.
 * @param		values	Values of the lookup table.
 * @param		numRptsPerTuple	Number of repeats per tuple.
 * @param		qrs		Query sequences.
 * @param		qryLen	Length of each query.
 * @param		maxQrySeqLen	Max query sequence length.
 * @param[out]	refIdx	Indices of the best matching reference sequences.
 * @param[out]	refPos	Positions of the best matching reference sequences.
 * @param		maxHits	Max number of hits.
 * @param		randNum	Random number.
 * @param		arrSize	Size of the arrays in shared memory.
 *
 * @note		Make sure the number of threads launched per block is less than
 * or equal to the number of elements in the arrays in the shared memory.
 */
__global__ void lookupTable6MapQry2_gpu(int *keys, int *values,
		int *numRptsPerTuple, char *qrs, uchar *qryLen, short maxQrySeqLen,
		char *refIdx, int *refPos, int maxHits, int seedLen, int randNum,
		int arrSize)
{
	int blockId = (blockIdx.y * gridDim.x) + blockIdx.x;
	short threadId = (threadIdx.y * blockDim.x) + threadIdx.x;
	long *refPos_shr = (long *) arr_shr;
	int *clusterSize_shr = (int *) &refPos_shr[arrSize];
	int *hash_shr = (int *) clusterSize_shr;
	int *randNums_shr = (int *) &clusterSize_shr[arrSize];
	short *bgstHits_shr = (short *) &randNums_shr[maxHits];
	short numQryTuples = qryLen[blockId] - seedLen + 1;
	randNumSeed = blockId + randNum;

	/* Initialize shared memory. */
	initializeShrMem_gpu(refPos_shr, clusterSize_shr, arrSize, threadId);

	/* Calculate query tuple hash. */
	short totalThreads = blockDim.x * blockDim.y * blockDim.z;
	short binSize = (short) ceil(((float) numQryTuples) / totalThreads);
	short i, threadId2 = threadId * binSize;
	int idx;
	for (i = 0; i < binSize; ++i)
	{
		if (threadId2 < numQryTuples)
		{
			idx = (blockId * maxQrySeqLen) + threadId2;
			hash_shr[threadId2] = getHash_gpu6(qrs + idx, seedLen);
		}
		++threadId2;
	}
	__syncthreads();

	/* Fetch query tuple hits from the global memory. */
	if (threadId == 0)
	{
		cpyHitsFromGlobalToShr_gpu(refPos_shr, hash_shr, arrSize, keys, values,
				numRptsPerTuple, numQryTuples, seedLen, maxQrySeqLen);
	}
	__syncthreads();

//	/* Sort query tuples (parallel merge-sort). */
//	sort_gpu6(refPos_shr, arrSize, threadId);
//
//	/* Create clusters, find biggest clusters, and choose randomly among
//	 * biggest clusters. */
//	if (threadId == 0)
//	{
//		short biggestClusterSize;
//		short numClusters = createClusters_gpu(refPos_shr, clusterSize_shr,
//				arrSize, &biggestClusterSize);
//		char numBiggestHits = findBiggestClusters_gpu(numClusters,
//				biggestClusterSize, clusterSize_shr, bgstHits_shr);
//		assignResults_gpu(blockId, maxHits, numBiggestHits, refPos_shr, refIdx,
//				refPos, bgstHits_shr, randNums_shr);
//	}
}


/**
 * This is a wrapper function that wraps @a cpyHitsFromGlobalToShr_gpu
 * function. It has been added so that @a cpyHitsFromGlobalToShr_gpu can be
 * unit-tested.
 *
 * @param[out]	refIdx	Reference indices.
 * @param[out]	shift	Shifts (reference position - query position).
 * @param[out]	refPos	Reference positions.
 * @param		hashes	Array containing tuple hash values for a query.
 * @param		arrSize	Number of elements in @a refIdx.
 * @param		keys	Array containing the indices to @a values.
 * @param		values	Array containing reference positions.
 * @param		numRptsPerTuple	Array containing number of repeats per reference
 * tuple.
 * @param		numQryTuples	Number of query tuples for a query.
 * @param		seedLen	Seed length.
 * @param		maxQryLen	Max query length.
 */
__global__ void cpyHitsFromGlobalToShr_gpu_wrap(long *refPos, int *hashes,
		int arrSize, int *keys, int *values, int *numRptsPerTuple,
		short numQryTuples, int seedLen, short maxQryLen)
{
	cpyHitsFromGlobalToShr_gpu(refPos, hashes, arrSize, keys, values,
			numRptsPerTuple, numQryTuples, seedLen, maxQryLen);
}


/**
 * Copies tuple hits from global memory to shared memory.
 *
 * @param[out]	refPos	Reference positions.
 * @param		hashes	Array containing tuple hash values for a query.
 * @param		arrSize	Number of elements in @a refIdx.
 * @param		keys	Array containing the indices to @a values.
 * @param		values	Array containing reference positions.
 * @param		numRptsPerTuple	Array containing number of repeats per reference
 * tuple.
 * @param		numQryTuples	Number of query tuples for a query.
 * @param		seedLen	Seed length.
 * @param		maxQryLen	Max query length.
 */
__device__ void cpyHitsFromGlobalToShr_gpu(long *refPos, int *hashes,
		int arrSize, int *keys, int *values, int *numRptsPerTuple,
		short numQryTuples, int seedLen, short maxQryLen)
{
	short numRefTuples = 0, i;
	int begin, end, hash, j, value;
	long tmp;
	for (i = 0; i < numQryTuples; ++i)
	{
		hash = hashes[i];
		begin = keys[hash];
		end = begin + numRptsPerTuple[hash] - 1;
		for (j = begin; j <= end; ++j)
		{
			value = values[j];
			tmp = value >> REF_POS_BITS2; /* Calculate reference index */
			refPos[numRefTuples] = tmp << REF_IDX_BITS2;
			tmp = (value & REF_POS_MASK2) * seedLen; /* Calculate position */
			refPos[numRefTuples] += tmp;
			tmp = tmp - i + maxQryLen; /* Calculate Shift */
			tmp = tmp << SHIFT_BITS2;
			refPos[numRefTuples] += tmp;
			++numRefTuples;

			/* If tuples cannot be fit into the shared memory, return. */
			if (numRefTuples >= arrSize)
				return;
		}
	}
}


/**
 * This is a wrapper function that wraps @a initializeShrMem_gpu function. It
 * has been added so that @a initializeShrMem_gpu can be unit-tested.
 *
 * @param[out]	refPos		Reference positions.
 * @param[out]	clusterSize	Cluster sizes.
 * @param		arrSize		Number of elements in @a refIdx.
 */
__global__ void intializeShrMem_gpu_wrap(long *refPos, int *clusterSize,
		int arrSize)
{
	short threadId = (threadIdx.y * blockDim.x) + threadIdx.x;
	initializeShrMem_gpu(refPos, clusterSize, arrSize, threadId);
}


/**
 * Initialize shared memory.
 *
 * @param[out]	refPos		Reference positions.
 * @param[out]	clusterSize	Cluster sizes.
 * @param		arrSize		Number of elements in @a refIdx.
 * @param		threadId	Thread ID.
 */
__device__ void initializeShrMem_gpu(long *refPos, int *clusterSize, int arrSize,
		short threadId)
{
	short totalThreads = blockDim.x * blockDim.y * blockDim.z;
	short binSize = (short) ceil(((float) (arrSize)) / totalThreads);
	short idx = threadId * binSize;
	short i;
	for (i = 0; i < binSize; ++i)
	{
		if (idx < arrSize)
		{
			refPos[idx] = REF_POS_MASK3;
			clusterSize[idx] = -1;
		}
		++idx;
	}
	__syncthreads();
}


/**
 * This is a wrapper function that wraps @a assignResults_gpu. It has been
 * added so that @a assignResults_gpu can be unit-tested.
 *
 * @param	maxHits		Max number of allowed hits.
 * @param	numBgstHits	Number of biggest hits.
 * @param	refIdx		Reference indices.
 * @param	refIdx_global	Reference index array on the global memory.
 * @param	refPos		Reference positions.
 * @param	refPos_global	Reference positions on the global memory.
 * @param	bgstClust	Array containing the indices of biggest clusters.
 * @param	randNum		Random number.
 */
__global__ void assignResults_gpu_wrap(char maxHits, char numBgstHits,
		long *refPos, char *refIdx_global, int *refPos_global, short *bgstClust,
		int randNum)
{
	randNumSeed = randNum;
	__shared__ int randNums[10];
	int blockId = (blockIdx.y * gridDim.x) + blockIdx.x;
	assignResults_gpu(blockId, maxHits, numBgstHits, refPos, refIdx_global,
			refPos_global, bgstClust, randNums);
}


/**
 * Assigns results to global memory.
 *
 * If the number of biggest clusters is greater than the maximum allowed
 * biggest clusters, then this function will randomly choose among all
 * the biggest clusters.
 *
 * @param	blockId		Current block ID.
 * @param	maxHits		Max number of allowed hits.
 * @param	numBgstHits	Number of biggest hits.
 * @param	refPos		Reference indices.
 * @param	refIdx_global	Reference index array on the global memory.
 * @param	refPos_global	Reference positions on the global memory.
 * @param	bgstClust	Array containing the indices of biggest clusters.
 * @param	randNums	Array that can be used for storing random numbers.
 */
__device__ void assignResults_gpu(int blockId, char maxHits, char numBgstHits,
		long *refPos, char *refIdx_global, int *refPos_global, short *bgstClust,
		int *randNums)
{
	int globalMemIdx = blockId * maxHits;
	char i;
	if (numBgstHits <= maxHits)
	{
		for (i = 0; i < numBgstHits; ++i)
		{
			refIdx_global[globalMemIdx + i] =
					(char) (refPos[bgstClust[i]] >> REF_IDX_BITS2);
			refPos_global[globalMemIdx + i] =
					(int) (refPos[bgstClust[i]] & REF_POS_MASK4);
		}
	}
	else
	{
		/* Randomly choose the required number of hits from among all the
		 * hits. */
		arrGetRandomNums_gpu6(maxHits, 0, numBgstHits - 1, randNums);
		for (i = 0; i < maxHits; ++i)
		{
			refIdx_global[globalMemIdx + i] =
					(char) (refPos[bgstClust[randNums[i]]] >> REF_IDX_BITS2);
			refPos_global[globalMemIdx + i] =
					(int) (refPos[bgstClust[randNums[i]]] & REF_POS_MASK4);
		}
	}
}


/**
 * This is a wrapper function that wraps @a findBiggestClusters_gpu function.
 * It has been added so that @a findBiggestClusters_gpu can be unit-tested.
 *
 * @param		numClust		Number of clusters.
 * @param		bgstClustSize	Biggest cluster size.
 * @param		clusterSize		Array containing cluster sizes.
 * @param[out]	bgstClust		Array containing biggest clusters.
 */
__global__ void findBiggestClusters_gpu_wrap(short numClust,
		short bgstClustSize, int *clusterSize, short *bgstClust,
		char *numBgstClust)
{
	*numBgstClust = findBiggestClusters_gpu(numClust, bgstClustSize,
			clusterSize, bgstClust);
}


/**
 * Find biggest clusters.
 *
 * @param		numClust		Number of clusters.
 * @param		bgstClustSize	Biggest cluster size.
 * @param		clusterSize		Array containing cluster sizes.
 * @param[out]	bgstClust		Array containing biggest clusters.
 * @return		Number of biggest clusters.
 */
__device__ char findBiggestClusters_gpu(short numClust, short bgstClustSize,
		int *clusterSize, short *bgstClust)
{
	char numBiggestHits = 0;
	short arrIdx = 0, i;
	for (i = 0; i < numClust; ++i)
	{
		if (bgstClustSize == clusterSize[i])
		{
			bgstClust[numBiggestHits] = arrIdx;
			++numBiggestHits;
		}
		arrIdx += clusterSize[i];
	}
	return numBiggestHits;
}


/**
 * This is a wrapper function that wraps @a createClusters_gpu function. It
 * has been added so that @a createClusters_gpu can be unit-tested.
 *
 * @param		refPos			Reference positions.
 * @param[out]	clusterSize		Cluster sizes.
 * @param		arrSize			Size of @a refIdx array.
 * @param[out]	bgstClustSize	Biggest cluster size.
 * @param[out]	numClusters		Number of clusters.
 */
__global__ void createClusters_gpu_wrap(long *refPos, int *clusterSize,
		int arrSize, short *bgstClustSize, short *numClusters)
{
	*numClusters = createClusters_gpu(refPos, clusterSize, arrSize,
			bgstClustSize);
}


/**
 * Creates clusters and returns the number of clusters.
 *
 * @note This function assumes that the input arrays are already sorted, first
 * by reference index and then by shift.
 *
 * @param		refPos			Reference positions.
 * @param[out]	clusterSize		Cluster sizes.
 * @param		arrSize			Size of @a refIdx array.
 * @param[out]	bgstClustSize	Biggest cluster size.
 * @return		Number of clusters.
 */
__device__ short createClusters_gpu(long *refPos, int *clusterSize, int arrSize,
		short *bgstClustSize)
{
	short numClusters = 0, i;
	clusterSize[numClusters] = 1;
	++numClusters;
	*bgstClustSize = 1;
	char refIdx;
	long pos1, pos2;
	for (i = 1; i < arrSize; ++i)
	{
		refIdx = (char) (refPos[i] >> REF_IDX_BITS2);
		if (refIdx == CHAR_MAX)
			break;
		pos1 = refPos[i - 1] >> SHIFT_BITS;
		pos2 = refPos[i] >> SHIFT_BITS;
		if (pos1 == pos2)
			++clusterSize[numClusters - 1];
		else
		{
			*bgstClustSize = max(*bgstClustSize, clusterSize[numClusters - 1]);
			clusterSize[numClusters] = 1;
			++numClusters;
		}
	}
	*bgstClustSize = max(*bgstClustSize, clusterSize[numClusters - 1]);
	return numClusters;
}


/**
 * This is a wrapper function that wraps @a sort_gpu6 function. It has been
 * added so that @a sort_gpu6 can be unit-tested.
 *
 * @param[in,out]	refPos			Reference positions.
 * @param			arrSize			Size of @a refIdx array.
 */
__global__ void sort_gpu_wrap6(long *refPos, int arrSize)
{
	short threadId = (threadIdx.y * blockDim.x) + threadIdx.x;
	sort_gpu6(refPos, arrSize, threadId);
}


/**
 * Sorts the given arrays in-place using parallel merge-sort. It first sorts
 * using the reference index, then shift, and finally by reference position.
 *
 * @param[in,out]	refPos			Reference positions.
 * @param			arrSize			Size of @a refIdx array.
 * @param			threadId		Current thread ID.
 */
__device__ void sort_gpu6(long *refPos, int arrSize, short threadId)
{
	short numThreads = blockDim.x * blockDim.y * blockDim.z;
	short binSize = (short) ceil(((float) arrSize) / numThreads);
	if (binSize < 2)
	{
		numThreads = numThreads / 2;
		binSize = (short) ceil(((float) arrSize) / numThreads);
	}
	int numBins = (int) ceil(((float) arrSize) / binSize);
	char isSorted;
	short i, begin, end;
	int tmp = 0;
	long tmpLong;
	while (numBins >= 0)
	{
		begin = (threadId * binSize) + (tmp % 2);
		end = min(begin + binSize - 1, arrSize - 1);
		isSorted = FALSE;
		while (isSorted == FALSE)
		{
			isSorted = TRUE;
			for (i = begin + 1; i <= end; ++i)
			{
				if (refPos[i] < refPos[i - 1])
				{
					tmpLong = refPos[i];
					refPos[i] = refPos[i - 1];
					refPos[i - 1] = tmpLong;
					isSorted = FALSE;
				}
			}
		}
		++tmp;
		--numBins;
		__syncthreads();
	}
}


/**
 * This is a wrapper function that wraps @a getHash_gpu6 function. It has been
 * added so that @a getHash_gpu6 can be unit-tested.
 *
 * @param		str		String for which hash is to be calculated.
 * @param		len		Length of @a str.
 * @param[out]	hash	Hash value of @a str.
 */
__global__ void getHash_gpu_wrap6(char *str, int len, int *hash)
{
	*hash = getHash_gpu6(str, len);
}


/**
 * Returns the hash value of the given string.
 *
 * @param	str	String for which hash is to be calculated.
 * @param	len	Length of @a str.
 * @return	Hash value.
 */
__device__ int getHash_gpu6(char *str, int len)
{
	int hash = 0, i;
	for (i = 0; i < len; ++i)
		hash += powerVals_gpu[i] * hashCodes_gpu[str[i]];
	return hash;
}


/**
 * Returns the index of the given number if it is already present in the
 * given array; otherwise, returns -1.
 *
 * @param	arr		Array in which the number will be searched.
 * @param	arrSize	Number of elements in the array. This number must be
 * equal to the number of elements in @a arr, otherwise it may result in invalid
 * behavior.
 * @param	num		The number to be searched in the array.
 * @return 	Index of the searched number in the array; otherwise, -1.
 */
__device__ int arrSearch_gpu6(int *arr, int arrSize, int num)
{
	int i = 0;
	for (i = 0; i < arrSize; ++i)
	{
		if (arr[i] == num)
			return i;
	}
	return -1;
}


/**
 * Fetches the given number of random numbers between the given limits.
 *
 * @param		n		Number of random numbers to be created. This number
 * should not be greater than the range specified by @a lowerLimit and
 * @a upperLimit.
 * @param		lLimit	The lower limit of the random numbers.
 * @param		uLimit	The upper limit of the random numbers.
 * @param[out] 	arr 	Array in which the random numbers will be stored.
 * The size of the array should be atleast @a n.
 */
__device__ void arrGetRandomNums_gpu6(int n, int lLimit, int uLimit, int *arr)
{
	int range = uLimit - lLimit + 1;
	int i = 0;
	int randNum;
	while (i < n)
	{
		randNum = (getRandNum_gpu6() % range) + lLimit;
		if (arrSearch_gpu6(arr, i, randNum) == -1)
		{
			arr[i] = randNum;
			++i;
		}
	}
}


/**
 * Returns a pseudo-random number.
 *
 * @return	Random number.
 *
 * @note This algorithm has been taken from "The C Programming Language"
 * by Kernighan and Ritchie.
 */
__device__ int getRandNum_gpu6()
{
	randNumSeed = (randNumSeed * 1103515245) + 12345;
	return ((randNumSeed / 65536) % 32768);
}


/**
 * Calculates the value of base raised to the n-th power.
 *
 * @param 	base	Base value.
 * @param 	n		Exponent value.
 * @return			The calculated value.
 */
__device__ int pow_gpu6(int base, int n)
{
	int p = 1;
	while (n > 0)
	{
		p = p * base;
		--n;
	}
	return p;
}
