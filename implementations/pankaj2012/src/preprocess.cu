#include "hip/hip_runtime.h"
/*
 * Copyright 2012, St. Jude Children's Research Hospital.
 * Written by Pankaj Gupta, pankaj.gupta@stjude.org.
 *
 * This file is part of Swift.  Swift is free software:  you can redistribute
 * it and/or modify it under the terms of the GNU General Public License as
 * published by the Free Software Foundation, either version 2 of the License,
 * or (at your option) any later version.
 *
 * Swift is distributed in the hope that it will be useful, but WITHOUT ANY
 * WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
 * A PARTICULAR PURPOSE.  See the GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along
 * with Swift.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <stdio.h>
#include "preprocess.h"
#include "common.h"


static int *_hashCodes = NULL; /* Array containing hash codes. */
static int *_powerVals = NULL; /* Array containing power values. */


/**
 * Creates and initializes data structures in this file.
 */
void preprocessCreate()
{
	_hashCodes = (int *) calloc(NUM_ASCII_CHARS, sizeof(int));
	_hashCodes['A'] = CODE_A;
	_hashCodes['a'] = CODE_A;
	_hashCodes['C'] = CODE_C;
	_hashCodes['c'] = CODE_C;
	_hashCodes['G'] = CODE_G;
	_hashCodes['g'] = CODE_G;
	_hashCodes['T'] = CODE_T;
	_hashCodes['t'] = CODE_T;
	_hashCodes['N'] = CODE_N;
	_hashCodes['n'] = CODE_N;

	_powerVals = (int *) calloc(MAX_SEED_LENGTH, sizeof(int));
	int i;
	for (i = 0; i < MAX_SEED_LENGTH; ++i)
		_powerVals[i] = (int) pow((float) 4, i);
}


/**
 * Deletes data structures in this file.
 */
void preprocessDelete()
{
	free(_hashCodes);
	_hashCodes = NULL;
	free(_powerVals);
	_powerVals = NULL;
}


/**
 * Returns the hash value of the given string
 *
 * @param s Given string
 * @param length Length of the given string
 * @return Hash value
 */
int getHash(char *s, int length)
{
	int i, sum;
	sum = 0;

	/* This section of code encodes the string into an integer.
	 * We use '4' as the base in the power function below because we
	 * have only 4 letters in our alphabet, i.e. A, C, G, and T.
	 *
	 * Example: Encoded value 'ACG' will be:
	 * = (4^0 * CODE_A) + (4^1 * CODE_C) + (4^2 * CODE_G)
	 * = (1 * CODE_A) + (4 * CODE_C) + (16 * CODE_G)
	 * (Assume CODE_A = 0, CODE_C = 1, and CODE_G = 2)
	 * = (1 * 0) + (4 * 1) + (16 * 2)
	 * = 0 + 4 + 32
	 * = 36
	 * That is, hash(ACG) = 36 */
	for (i = 0; i < length; ++i)
		sum += _powerVals[i] * _hashCodes[s[i]];

	return sum;
}


/**
 * Returns the hash value of the given string
 *
 * @param s Given string
 * @param length Length of the given string
 * @return Hash value
 */
long getLongHash(char *s, int length)
{
	int i;
	long sum = 0;

	/* This section of code encodes the string into an integer.
	 * We use '4' as the base in the power function below because we
	 * have only 4 letters in our alphabet, i.e. A, C, G, and T.
	 *
	 * Example: Encoded value 'ACG' will be:
	 * = (4^0 * CODE_A) + (4^1 * CODE_C) + (4^2 * CODE_G)
	 * = (1 * CODE_A) + (4 * CODE_C) + (16 * CODE_G)
	 * (Assume CODE_A = 0, CODE_C = 1, and CODE_G = 2)
	 * = (1 * 0) + (4 * 1) + (16 * 2)
	 * = 0 + 4 + 32
	 * = 36
	 * That is, hash(ACG) = 36 */
	for (i = 0; i < length; ++i)
		sum += _powerVals[i] * _hashCodes[s[i]];

	return sum;
}



///**
// * Returns the hash value of the given string
// *
// * @param s Given string
// * @param length Length of the given string
// * @return Hash value
// */
//int getHash(char *s, int length)
//{
//	static int i, sum;
//	sum = 0;
//
//	/* This section of code encodes the string into an integer.
//	 * We use '4' as the base in the power function below because we
//	 * have only 4 letters in our alphabet, i.e. A, C, G, and T.
//	 *
//	 * Example: Encoded value 'ACG' will be:
//	 * = (4^0 * CODE_A) + (4^1 * CODE_C) + (4^2 * CODE_G)
//	 * = (1 * CODE_A) + (4 * CODE_C) + (16 * CODE_G)
//	 * (Assume CODE_A = 0, CODE_C = 1, and CODE_G = 2)
//	 * = (1 * 0) + (4 * 1) + (16 * 2)
//	 * = 0 + 4 + 32
//	 * = 36
//	 * That is, hash(ACG) = 36 */
//	for (i = 0; i < length; ++i)
//	{
//		if (s[i] == 'A' || s[i] == 'a')
//			sum += (int) pow((float) 4, i) * CODE_A;
//		else if (s[i] == 'C' || s[i] == 'c')
//			sum += (int) pow((float) 4, i) * CODE_C;
//		else if (s[i] == 'G' || s[i] == 'g')
//			sum += (int) pow((float) 4, i) * CODE_G;
//		else if (s[i] == 'T' || s[i] == 't')
//			sum += (int) pow((float) 4, i) * CODE_T;
//		else if (s[i] == 'N' || s[i] == 'n')
//			sum += (int) pow((float) 4, i) * CODE_N;
//		else if (s[i] == '*')
//			sum += (int) pow((float) 4, i) * CODE_STAR;
//		else
//			sum += 0;
//	}
//
//	return sum;
//}


/**
 * Decodes the given hash
 *
 * @param hash Hash value to be decoded
 * @param[out] s Decoded sequence
 * @param seedLength Seed length
 */
void getUnhash(int hash, char *s, int seedLength)
{
	int quotient = hash, remainder, i = -1;

	while (quotient > 0)
	{
		remainder = quotient % 4;
		++i;
		if (remainder == 0)
			s[i] = 'A';
		else if (remainder == 1)
			s[i] = 'C';
		else if (remainder == 2)
			s[i] = 'G';
		else if (remainder == 3)
			s[i] = 'T';
		else
			s[i] = 'N';
		quotient = quotient / 4;
	}

	/* Pad with 'A's */
	while (i < (seedLength - 1))
	{
		++i;
		s[i] = 'A';
	}
	s[seedLength] = '\0';
}
