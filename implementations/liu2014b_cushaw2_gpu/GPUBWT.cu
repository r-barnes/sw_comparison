#include "hip/hip_runtime.h"
/*
 * GPUBWT.cu
 *
 *  Created on: Jan 7, 2013
 *      Author: yongchao
 */
#include "GPUBWT.h"
#include "GPUSeeds.h"
#include "GPUVariables.h"

GPUBWT::GPUBWT(uint32_t* hostBwtPtr, size_t bwtSize, uint32_t dollar,
		uint32_t seqLen, uint32_t* ccounts) {

	Utils::log("load BWT (%u MB)\n", bwtSize * sizeof(uint32_t) / 1048576);

	/*calculate the width and height*/
	uint32_t bwtWidthShift = 16;
	uint32_t bwtWidthMask = (1 << bwtWidthShift) - 1;
	hipMemcpyToSymbol(HIP_SYMBOL(_cudaBwtWidthShift), &bwtWidthShift, sizeof(uint32_t), 0,
			hipMemcpyHostToDevice);
	myCheckCudaError;

	hipMemcpyToSymbol(HIP_SYMBOL(_cudaBwtWidthMask), &bwtWidthMask, sizeof(uint32_t), 0,
			hipMemcpyHostToDevice);
	myCheckCudaError;

	/*allocate space for bwt data*/
	uint32_t bwtWidth = 1 << bwtWidthShift;
	uint32_t bwtHeight = (bwtSize + bwtWidthMask) >> bwtWidthShift;
	hipChannelFormatDesc channelDes = hipCreateChannelDesc<uint32_t>();
	hipMallocArray(&_bwtDevPtr, &channelDes, bwtWidth, bwtHeight);

	/*copy the data*/
	uint32_t* data;

	/*get a copy of the original data*/
	hipHostMalloc(&data, bwtWidth * bwtHeight * sizeof(uint32_t));
	myCheckCudaError;
	memcpy(data, hostBwtPtr, bwtSize * sizeof(uint32_t));

	/*copy to the CUDA array*/
	hipMemcpy2DToArray(_bwtDevPtr, 0, 0, data,
			bwtWidth * sizeof(uint32_t), bwtWidth * sizeof(uint32_t), bwtHeight,
			hipMemcpyHostToDevice);
	myCheckCudaError;

	/*release the temp data*/
	hipHostFree(data);
	myCheckCudaError;

	/*set texture parameters*/
	_texBWT.addressMode[0] = hipAddressModeClamp;
	_texBWT.addressMode[1] = hipAddressModeClamp;
	_texBWT.filterMode = hipFilterModePoint;
	_texBWT.normalized = false;

	/*bind the texture memory*/
	hipBindTextureToArray(_texBWT, _bwtDevPtr, channelDes);
	myCheckCudaError;

	/*other parameters*/
	hipMemcpyToSymbol(HIP_SYMBOL(_cudaBwtDollar), &dollar, sizeof(dollar), 0,
			hipMemcpyHostToDevice);
	myCheckCudaError;

	hipMemcpyToSymbol(HIP_SYMBOL(_cudaBwtSeqLength), &seqLen, sizeof(seqLen), 0,
			hipMemcpyHostToDevice);
	myCheckCudaError;

	hipMemcpyToSymbol(HIP_SYMBOL(_cudaBwtCCounts), ccounts, sizeof(uint32_t) * BWT_NUM_OCC,
			0, hipMemcpyHostToDevice);
	myCheckCudaError;
}

GPUBWT::~GPUBWT() {
	/*unbind texture memory*/
	hipUnbindTexture(_texBWT);
	myCheckCudaError;

	/*release device memory*/
	hipFreeArray(_bwtDevPtr);
	myCheckCudaError;
}
