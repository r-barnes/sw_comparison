#include "hip/hip_runtime.h"
#include "SuffixArray.h"
#include "Utils.h"
#ifdef __HIPCC__
#include "GPUMacros.h"
#endif

SuffixArray::SuffixArray(const char* saFileName)
{
	FILE* file;
	uint32_t intVar;

	//open the suffix array file
	file = fopen(saFileName, "rb");
	if (!file)
	{
		Utils::exit("Faield to open suffix array file: %s", saFileName);
	}

	//get the $ symbol position
	fread(&intVar, sizeof(uint32_t), 1, file);

	//skip
	fseek(file, 4 * sizeof(uint32_t), SEEK_CUR);

	//get the suffix array interval
	fread(&_saFactor, sizeof(uint32_t), 1, file);

	//get the sequence length
	fread(&_seqLength, sizeof(uint32_t), 1, file);

	//get the suffix array data size
	_saSize = (_seqLength + _saFactor) / _saFactor;
	if (_saSize < 1)
		_saSize = 1;

	//read the suffix array data
#ifndef __HIPCC__
	_saPtr = new uint32_t[_saSize];
#else
	hipHostAlloc(&_saPtr, _saSize * sizeof(uint32_t), hipHostMallocMapped);
	myCheckCudaError;
#endif
	_saPtr[0] = (uint32_t) -1;
	fread(_saPtr + 1, sizeof(uint32_t), _saSize - 1, file);

	//close the file
	fclose(file);

	//report the suffix array memory size
	Utils::log("Suffix array memory size: %g MB\n",
			(_saSize * sizeof(uint32_t)) / 1024.0 / 1024.0);
}
SuffixArray::~SuffixArray()
{
	if (_saPtr)
	{
#ifndef __HIPCC__
		delete[] _saPtr;
#else
		hipHostFree(_saPtr);
#endif
	}
}

uint32_t SuffixArray::getFactor(const char* saFileName)
{
	FILE* file;
	uint32_t intVar;

	//open the suffix array file
	file = fopen(saFileName, "rb");
	if (!file)
	{
		Utils::exit("Failed to open suffix array file: %s", saFileName);
	}

	//skip
	fseek(file, 5 * sizeof(uint32_t), SEEK_SET);

	//get the suffix array factor
	fread(&intVar, sizeof(uint32_t), 1, file);

	//close the file
	fclose(file);

	//return the suffix array factor
	return intVar;
}

