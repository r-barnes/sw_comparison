#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "CFastaSW.h"
#define CUERR do{ hipError_t err;			\
	if ((err = hipGetLastError()) != hipSuccess) {		\
			int device;	\
			hipGetDevice(&device);	\
  		fprintf(stderr, "CUDA error on GPU %d: %s : %s, line %d\n", device, hipGetErrorString(err), __FILE__, __LINE__); }}while(0);

CFastaSW::CFastaSW() {
	uchar_channelDesc = hipCreateChannelDesc<unsigned char>();
	uchar4_channelDesc = hipCreateChannelDesc<uchar4>();
	uint_channelDesc = hipCreateChannelDesc<unsigned int>();
	uint2_channelDesc = hipCreateChannelDesc<uint2>();
	uint4_channelDesc = hipCreateChannelDesc<uint4>();
	char4_channelDesc = hipCreateChannelDesc<char4>();
	sint_channelDesc = hipCreateChannelDesc<int>();
	sint4_channelDesc = hipCreateChannelDesc<int4>();
}
CFastaSW::~CFastaSW() {
	//do nothing
}
hipArray* CFastaSW::swMallocArray(int width, int height, int type) {
	hipArray* cu_array;

	switch (type) {
	case pChannelFormatKindUnsignedChar:
		hipMallocArray(&cu_array, &uchar_channelDesc, width, height);
		break;
	case pChannelFormatKindUnsignedChar4:
		hipMallocArray(&cu_array, &uchar4_channelDesc, width, height);
		break;
	case pChannelFormatKindUnsigned:
		hipMallocArray(&cu_array, &uint_channelDesc, width, height);
		break;
	case pChannelFormatKindUnsignedInt4:
		hipMallocArray(&cu_array, &uint4_channelDesc, width, height);
		break;
	case pChannelFormatKindUnsignedInt2:
		hipMallocArray(&cu_array, &uint2_channelDesc, width, height);
		break;
	case pChannelFormatKindChar4:
		hipMallocArray(&cu_array, &char4_channelDesc, width, height);
		break;
	case pChannelFormatKindSignedInt:
		hipMallocArray(&cu_array, &sint_channelDesc, width, height);
		break;
	case pChannelFormatKindSignedInt4:
		hipMallocArray(&cu_array, &sint4_channelDesc, width, height);
		break;
	default:
		fprintf(stderr, "Unknown cuda array type\n");
		exit(-1);
	}

	CUERR
	return cu_array;

}

void CFastaSW::transferResult(int numSeqs) {
	hipMemcpy(hostResult, cudaResult, numSeqs * sizeof(SeqEntry),
			hipMemcpyDeviceToHost);
	CUERR
	;
}
