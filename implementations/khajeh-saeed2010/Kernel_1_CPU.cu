////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on CPU.
// Straight accumulation in double precision.
////////////////////////////////////////////////////////////////////////////
 
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
using namespace std;


void Kernel_1_CPU(int *A, int *B, int *Max_CPU, int *A_Location_CPU, int *B_Location_CPU, 
			int K1_Max_Report, int LA, int LB, int sim, int dissim, int Gop, int Gex)					  

{
	int i,j,S1,H1,EF, H_Max	;

	int *F, *E, *H;
	F = new int [LA];
	E = new int [LA+1];
	H = new int [(LA+1)*2];
	//Max_CPU = new int [K1_Max_Report];

	for (j=0; j<LA; j++){
	F[j]=0;
	E[j]=0;
	}
	E[LA+1]=0;


	for (j=0; j<(2*(LA+1)); j++)
	H[j]=0;

	for (i=0; i<LB; i++)
	{
		for (j=0; j<LA; j++)
		{
			if (A[j]==B[i])
			S1=H[j]+sim;
			else 
			S1=H[j]+dissim;
			H1=max(S1,0);
			F[j]=max(F[j]-Gex,H[j+1]-Gop);
			E[j+1]=max(E[j]-Gex,H[(LA+1)+j]-Gop);
			EF=max(F[j],E[j+1]);
			H[(LA+1)+j+1]=max(EF,H1);
		}
		for (j=1; j<(LA+1); j++)
		{
			H[j]=H[(LA+1)+j];
			H_Max = H[j];

			if ((H_Max>1) && (H_Max>Max_CPU[0]))
			{
				if (A[j-1]==B[i])
				{
					Max_CPU[0]=H_Max;
					A_Location_CPU[0]=j-1;
					B_Location_CPU[0]=i;

					for (int k=0; k<K1_Max_Report-1; k++)
					{
						if (Max_CPU[k]>Max_CPU[k+1])
						{		
							int temp1=Max_CPU[k+1];
							int temp2=A_Location_CPU[k+1];
							int temp3=B_Location_CPU[k+1];	
							Max_CPU[k+1]=Max_CPU[k];
							A_Location_CPU[k+1]=A_Location_CPU[k];
							B_Location_CPU[k+1]=B_Location_CPU[k];
							Max_CPU[k]=temp1;
							A_Location_CPU[k]=temp2;
							B_Location_CPU[k]=temp3;
						}
						else
						{
							k=K1_Max_Report;
						}
					}
				}
			}
		}
	}
	for (int i=0; i<(K1_Max_Report-1)/2 + 1 ; i++)
	{
		int Temp = Max_CPU[K1_Max_Report-i-1];
		Max_CPU[K1_Max_Report-i-1]=Max_CPU[i];
		Max_CPU[i]=Temp;
	}
}

/*	 printf("No. Sim. Val.  Seq. A    Seq. B \n");
	 printf("-------------------------------- \n");
    for( i = 1; i < (K1_Max_Report+1); ++i)     
    {
   		 printf(" %i     %i       %i       %i \n", i, Max_CPU[K1_Max_Report-i],int(fmod(1.0*End_Point[K1_Max_Report-i],(LA+1))), int(1.0*(End_Point[K1_Max_Report-i])/(LA+1)));
    }  */



//		cout<<A[j]<<"-----"<<B[i]<<"-----"<<F[j]<<"------"<<E[j]<<endl;	